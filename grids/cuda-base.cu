#ifndef CUDA_BASE_H
#define CUDA_BASE_H

#include <stdexcept>
#include "util.cu"

/** Cuda Base Grid
 *
 * Provides allocation of memory and a struct to pass information about a grid
 * to kernels. Abstract basis for grids implemented in Cuda.
 */
template<typename value_t, typename coord_t>
class CudaBaseGrid : 
virtual public Grid<value_t, coord_t>
{
    public:
    
    CudaBaseGrid();
    ~CudaBaseGrid();    

    virtual void allocate();
    virtual void deallocate();
    virtual void fill(value_t v);
    //virtual void fill(double v);
    //virtual void fill(float v);


    /** A *blocking* call that synchronizes the current state of the grid to
     * the device or host. Call before host/device accesses to prevent page
     * faults inside the kernel. */
    void prefetch(int device = -1);
    void prefetchToDevice();
    void prefetchToHost();

    void setSmemBankSize(int sz=-1);

};

template<typename value_t, typename coord_t>
CudaBaseGrid<value_t, coord_t>::CudaBaseGrid() { }

template<typename value_t, typename coord_t>
CudaBaseGrid<value_t, coord_t>::~CudaBaseGrid() {
    if(this->data) {
        this->deallocate();
    }
}

template<typename value_t, typename coord_t>
void CudaBaseGrid<value_t, coord_t>::allocate() {
    value_t *ptr;
    CUDA_THROW( hipMallocManaged(&ptr, this->size) );
    CUDA_THROW( hipMemset(ptr, 0, this->size) );
    this->setSmemBankSize();
    this->data = ptr;
}

template<typename value_t, typename coord_t>
void CudaBaseGrid<value_t, coord_t>::setSmemBankSize(int sz) {
    if(sz == -1) {
        sz = sizeof(value_t);
    }
    if (sz == 4) {
        CUDA_THROW( hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte) );
    } else if(sz == 8) {
        CUDA_THROW( hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte) );
    }
}

template<typename value_t, typename coord_t>
void CudaBaseGrid<value_t, coord_t>::deallocate() {
    if(this->data) {
        CUDA_THROW( hipFree(this->data) );
        this->data = NULL;
    }
}

template<typename value_t, typename coord_t>
void CudaBaseGrid<value_t, coord_t>::fill(value_t v) {
    CUDA_THROW( hipMemset(this->data, 0, this->size) );
}

template<>
void CudaBaseGrid<double, coord3>::fill(double v) {
    CUDA_THROW( hipMemset(this->data, v, this->size) );
}

template<>
void CudaBaseGrid<float, coord3>::fill(float v) {
    CUDA_THROW( hipMemset(this->data, v, this->size) );
}

template<typename value_t, typename coord_t>
void CudaBaseGrid<value_t, coord_t>::prefetch(int device) {
    if(device == -1) {
        CUDA_THROW( hipGetDevice(&device) );
    }
    CUDA_THROW( hipMemPrefetchAsync(this->data, this->size, device, 0) );
    CUDA_THROW( hipDeviceSynchronize() );
}


template<typename value_t, typename coord_t>
void CudaBaseGrid<value_t, coord_t>::prefetchToDevice() {
    this->prefetch();
}

template<typename value_t, typename coord_t>
void CudaBaseGrid<value_t, coord_t>::prefetchToHost() {
    this->prefetch(hipCpuDeviceId);
}

#endif