#include "hip/hip_runtime.h"
#ifndef COORD3_GRID_H
#define COORD3_GRID_H
#include <stdio.h>
#include "grid.cu"
#include "coord3.cu"

/** This is still a fairly useless grid as it does not commit on a memory
 * layout and thus does not provide access to storing or getting data.
 * It does, however, provide some functionality that will be common to any grid
 * that uses the coord3 data type for coordinates.
 */

template<typename value_t>
class Coord3BaseGrid : 
virtual public Grid<value_t, coord3> {

    public:
    
    Coord3BaseGrid();

    /** Load values from another grid into this one, if coordinate and value
     * types are compatible. */
    //template<typename other_value_t, typename other_allocator_t>
    void import(Grid<value_t, coord3> *other) final;

    /** Fill all the cells in the grid with the same value. */
    void fill(value_t value);

    /** Print the grid (makes sense for small sizes for debugging). */
    void print();

};

//  IMPLEMENTATIONS

template<typename value_t>
Coord3BaseGrid<value_t>::Coord3BaseGrid() {}

//template<typename value_t, typename allocator, typename other_value_t, typename other_allocator_t>
template<typename value_t>
void Coord3BaseGrid<value_t>::import(Grid<value_t, coord3> *other) {
    int N = std::min(this->dimensions.x, other->dimensions.x);
    int M = std::min(this->dimensions.y, other->dimensions.y);
    int L = std::min(this->dimensions.z, other->dimensions.z);
    for(int x = 0; x < N; x++) {
        for(int y = 0; y < M; y++) {
            for(int z = 0; z < L; z++) {
                this->set(coord3(x, y, z), (*other)[coord3(x, y, z)]);
            }
        }
    }
}

template<typename value_t>
void Coord3BaseGrid<value_t>::fill(value_t value) {
    int N = this->dimensions.x;
    int M = this->dimensions.y;
    int L = this->dimensions.z;
    for(int x = 0; x < N; x++) {
        for(int y = 0; y < M; y++) {
            for(int z = 0; z < L; z++) {
                this->set(coord3(x, y, z), value);
            }
        }
    }
}

template<typename value_t>
void Coord3BaseGrid<value_t>::print() {
    int N = this->dimensions.x;
    int M = this->dimensions.y;
    int L = this->dimensions.z;
    for(int x=0; x<N; x++) {
        for(int y=0; y<M; y++) {
            printf("[");
            for(int z=0; z<L; z++) {
                printf("%5.1f", (*this)[coord3(x, y, z)]);
            }
            printf("]  ");
        }
        printf("\n");
    }
}

#endif