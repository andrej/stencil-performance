#include "hip/hip_runtime.h"
template<typename value_t>
__global__
void laplap_idxvar(GRID_ARGS const coord3 halo, const coord3 max_coord, const value_t *in, value_t *out) {
    const int i = blockIdx.x*blockDim.x + threadIdx.x + halo.x;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + halo.y;
    const int k = blockIdx.z*blockDim.z + threadIdx.z + halo.z;
    if(i >= max_coord.x || j >= max_coord.y || k >= max_coord.z) {
        return;
    }

    const int center        = INDEX(i, j, k);
    const int left          = NEIGHBOR_OF_INDEX(center, -1,  0, 0);
    const int leftleft      = NEIGHBOR_OF_INDEX(  left, -1,  0, 0);
    const int topleft       = NEIGHBOR_OF_INDEX(  left,  0, -1, 0);
    const int bottomleft    = NEIGHBOR_OF_INDEX(  left,  0, +1, 0);

    const int right         = NEIGHBOR_OF_INDEX(center, +1,  0, 0);
    const int topright      = NEIGHBOR_OF_INDEX( right,  0, -1, 0);
    const int rightright    = NEIGHBOR_OF_INDEX( right, +1,  0, 0);
    const int bottomright   = NEIGHBOR_OF_INDEX( right,  0, +1, 0);

    const int top           = NEIGHBOR_OF_INDEX(center,  0, -1, 0);
    const int toptop        = NEIGHBOR_OF_INDEX(   top,  0, -1, 0);

    const int bottom        = NEIGHBOR_OF_INDEX(center,  0, +1, 0);
    const int bottombottom  = NEIGHBOR_OF_INDEX(bottom,  0, +1, 0);


    const value_t lap_center= 4 * in[center]
                                - in[left]
                                - in[right]
                                - in[top]
                                - in[bottom];
    const value_t lap_left  = 4 * in[left]
                                - in[leftleft]
                                - in[center]
                                - in[topleft]
                                - in[bottomleft];
    const value_t lap_right = 4 * in[right]
                                - in[center]
                                - in[rightright]
                                - in[topright]
                                - in[bottomright];
    const value_t lap_top   = 4 * in[top]
                                - in[topleft]
                                - in[topright]
                                - in[toptop]
                                - in[center];
    const value_t lap_bottom= 4 * in[bottom]
                                - in[bottomleft]
                                - in[bottomright]
                                - in[center]
                                - in[bottombottom];
    out[center]             = 4 * lap_center
                                - lap_left
                                - lap_right
                                - lap_top
                                - lap_bottom;
}