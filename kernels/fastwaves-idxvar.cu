#include "hip/hip_runtime.h"
template<typename value_t>
__global__
void fastwaves_idxvar(const FastWavesBenchmark::Info info,
                      GRID_ARGS
                      const value_t *ppuv,
                      const value_t *wgtfac,
                      const value_t *hhl,
                      const value_t *vin,
                      const value_t *uin,
                      const value_t *vtens,
                      const value_t *utens, 
                      const value_t *rho,
                      const value_t *fx,
                      const double edadlat,
                      const double dt_small,
                      const int c_flat_limit,
                      value_t *uout,
                      value_t *vout) {
    const int i = blockIdx.x*blockDim.x + threadIdx.x + info.halo.x;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + info.halo.y;
    const int k = blockIdx.z*blockDim.z + threadIdx.z + info.halo.z;
    if(i >= info.max_coord.x || j >= info.max_coord.y || k >= info.max_coord.z - 1) {
        return;
    }

    const int idx_0_0_n1  = NEIGHBOR(i, j, k, 0, 0, -1);
    const int idx_0_0_0   = NEXT_Z_NEIGHBOR_OF_INDEX(idx_0_0_n1);
    const int idx_0_0_p1  = NEXT_Z_NEIGHBOR_OF_INDEX(idx_0_0_0);
    const int idx_p1_0_n1 = NEIGHBOR_OF_INDEX(idx_0_0_n1, +1, 0, 0);
    const int idx_p1_0_0  = NEXT_Z_NEIGHBOR_OF_INDEX(idx_p1_0_n1);
    const int idx_p1_0_p1 = NEXT_Z_NEIGHBOR_OF_INDEX(idx_p1_0_0);
    const int idx_0_p1_n1 = NEIGHBOR_OF_INDEX(idx_0_0_n1, 0, +1, 0);
    const int idx_0_p1_0  = NEXT_Z_NEIGHBOR_OF_INDEX(idx_0_p1_n1);
    const int idx_0_p1_p1 = NEXT_Z_NEIGHBOR_OF_INDEX(idx_0_p1_0);

    // ppgu, ppgv
    value_t ppgu, ppgv;
    if(k < c_flat_limit + info.halo.z) {
        ppgu = ppuv[idx_p1_0_0] - ppuv[idx_0_0_0];
        ppgv = ppuv[idx_0_p1_0] - ppuv[idx_0_0_0];
    } else {
        value_t ppgk_0_0_0, ppgk_p1_0_0, ppgk_0_p1_0, ppgk_0_0_p1, ppgk_p1_0_p1, ppgk_0_p1_p1;
        value_t ppgc_0_0_0, ppgc_p1_0_0, ppgc_0_p1_0;
        ppgk_0_0_0   = wgtfac[idx_0_0_0] * ppuv[idx_0_0_0] +
                        (1.0 - wgtfac[idx_0_0_0]) * ppuv[idx_0_0_n1];
        ppgk_p1_0_0  = wgtfac[idx_p1_0_0] * ppuv[idx_p1_0_0] +
                        (1.0 - wgtfac[idx_p1_0_0]) * ppuv[idx_p1_0_n1];
        ppgk_0_p1_0  = wgtfac[idx_0_p1_0] * ppuv[idx_0_p1_0] +
                        (1.0 - wgtfac[idx_0_p1_0]) * ppuv[idx_0_p1_n1];
        ppgk_0_0_p1  = wgtfac[idx_0_0_p1] * ppuv[idx_0_0_p1] +
                        (1.0 - wgtfac[idx_0_0_p1]) * ppuv[idx_0_0_0];
        ppgk_p1_0_p1 = wgtfac[idx_p1_0_p1] * ppuv[idx_p1_0_p1] +
                        (1.0 - wgtfac[idx_p1_0_p1]) * ppuv[idx_p1_0_0];
        ppgk_0_p1_p1 = wgtfac[idx_0_p1_p1] * ppuv[idx_0_p1_p1] +
                        (1.0 - wgtfac[idx_0_p1_p1]) * ppuv[idx_0_p1_0];
        ppgc_0_0_0  = ppgk_0_0_p1  - ppgk_0_0_0;
        ppgc_p1_0_0 = ppgk_p1_0_p1 - ppgk_p1_0_0;
        ppgc_0_p1_0 = ppgk_0_p1_p1 - ppgk_0_p1_0;
        ppgu =
            (ppuv[idx_p1_0_0] - ppuv[idx_0_0_0]) + (ppgc_p1_0_0 + ppgc_0_0_0) * 0.5 * 
            ((hhl[idx_0_0_p1] + hhl[idx_0_0_0]) - (hhl[idx_p1_0_p1] + hhl[idx_p1_0_0])) / 
            ((hhl[idx_0_0_p1] - hhl[idx_0_0_0]) + (hhl[idx_p1_0_p1] - hhl[idx_p1_0_0]));
        ppgv =
            (ppuv[idx_0_p1_0] - ppuv[idx_0_0_0]) + (ppgc_0_p1_0 + ppgc_0_0_0) * 0.5 *
            ((hhl[idx_0_0_p1] + hhl[idx_0_0_0]) - (hhl[idx_0_p1_p1] + hhl[idx_0_p1_0])) /
            ((hhl[idx_0_0_p1] - hhl[idx_0_0_0]) + (hhl[idx_0_p1_p1] - hhl[idx_0_p1_0]));
    }

    // out
    uout[idx_0_0_0] =
        uin[idx_0_0_0] + dt_small * (utens[idx_0_0_0] - ppgu * 
        (fx[idx_0_0_0] / (0.5 * (rho[idx_p1_0_0] + rho[idx_0_0_0]))));
    vout[idx_0_0_0] =
        vin[idx_0_0_0] + dt_small * (vtens[idx_0_0_0] - ppgv * 
        (edadlat / (0.5 * ((rho[idx_0_p1_0] + rho[idx_0_0_0])))));
}