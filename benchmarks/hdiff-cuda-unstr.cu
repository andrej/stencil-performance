#include "hip/hip_runtime.h"
#ifndef HDIFF_CUDA_UNSTR_H
#define HDIFF_CUDA_UNSTR_H
#include "benchmarks/benchmark.cu"
#include "coord3.cu"
#include "grids/grid.cu"
#include "grids/cuda-regular.cu"
#include "grids/cuda-unstructured.cu"

namespace HdiffCudaUnstr {

    /** Variants of this benchmark. */
    enum Variant { RegNaive, RegKloop, UnstrNaive, UnstrKloop, UnstrIdxvars };

    /** Information about this benchmark for use in the kernels. */
    struct Info {
        coord3 halo;
        coord3 inner_size;
    };

    /** Naive implementation of a unstructured grid horizontal diffusion
     * kernel. Runs index calculations in every k-iteration.
     */
     __global__
     void kernel_naive(Info info,
                         CudaUnstructuredGrid3DInfo<double> in,
                         CudaUnstructuredGrid3DInfo<double> out,
                         CudaUnstructuredGrid3DInfo<double> coeff
                         #ifdef HDIFF_DEBUG
                         , CudaUnstructuredGrid3DInfo<double> dbg_lap
                         , CudaUnstructuredGrid3DInfo<double> dbg_flx
                         , CudaUnstructuredGrid3DInfo<double> dbg_fly
                         #endif
                         ) {
        const int i = threadIdx.x + blockIdx.x*blockDim.x + info.halo.x;
        const int j = threadIdx.y + blockIdx.y*blockDim.y + info.halo.y;
        const int k = threadIdx.z + blockIdx.z*blockDim.z + info.halo.z;
        if(i-info.halo.x >= info.inner_size.x || j-info.halo.y >= info.inner_size.y || k-info.halo.z >= info.inner_size.z) {
            return;
        }
        coord3 coord(i, j, k);
        
        int n_0_0_0       = CUDA_UNSTR_INDEX(in, coord);
        int n_0_n1_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_0_0,   0, -1, 0);
        int n_0_n2_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_n1_0,  0, -1, 0);
        int n_n1_0_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_0_0,  -1, 0, 0);
        int n_n1_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_n1_0_0,  0, -1, 0);
        int n_n2_0_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_n1_0_0, -1, 0, 0);
        //int n_n2_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_n2_0_0,  0, -1, 0);
        int n_0_p1_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_0_0,   0, +1, 0);
        int n_0_p2_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_p1_0,  0, +1, 0);
        int n_p1_0_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_0_0,  +1, 0, 0);
        int n_p1_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_p1_0_0,  0, +1, 0);
        int n_p2_0_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_p1_0_0, +1, 0, 0);
        //int n_p2_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_p2_0_0,  0, +1, 0);     
        int n_n1_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_n1_0_0,  0, +1, 0);
        int n_p1_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_p1_0_0,  0, -1, 0);

        double lap_ij = 
            4 * CUDA_UNSTR_AT(in, n_0_0_0) 
            - CUDA_UNSTR_AT(in, n_n1_0_0) - CUDA_UNSTR_AT(in, n_p1_0_0)
            - CUDA_UNSTR_AT(in, n_0_n1_0) - CUDA_UNSTR_AT(in, n_0_p1_0);
        double lap_imj = 
            4 * CUDA_UNSTR_AT(in, n_n1_0_0)
            - CUDA_UNSTR_AT(in, n_n2_0_0) - CUDA_UNSTR_AT(in, n_0_0_0)
            - CUDA_UNSTR_AT(in, n_n1_n1_0) - CUDA_UNSTR_AT(in, n_n1_p1_0);
        double lap_ipj =
            4 * CUDA_UNSTR_AT(in, n_p1_0_0)
            - CUDA_UNSTR(in, coord) - CUDA_UNSTR_AT(in, n_p2_0_0)
            - CUDA_UNSTR_AT(in, n_p1_n1_0) - CUDA_UNSTR_AT(in, n_p1_p1_0);
        double lap_ijm =
            4 * CUDA_UNSTR_AT(in, n_0_n1_0)
            - CUDA_UNSTR_AT(in, n_n1_n1_0) - CUDA_UNSTR_AT(in, n_p1_n1_0)
            - CUDA_UNSTR_AT(in, n_0_n2_0) - CUDA_UNSTR(in, coord);
        double lap_ijp =
            4 * CUDA_UNSTR_AT(in, n_0_p1_0)
            - CUDA_UNSTR_AT(in, n_n1_p1_0) - CUDA_UNSTR_AT(in, n_p1_p1_0)
            - CUDA_UNSTR(in, coord) - CUDA_UNSTR_AT(in, n_0_p2_0);

        double flx_ij = lap_ipj - lap_ij;
        flx_ij = flx_ij * (CUDA_UNSTR_AT(in, n_p1_0_0) - CUDA_UNSTR(in, coord)) > 0 ? 0 : flx_ij;

        double flx_imj = lap_ij - lap_imj;
        flx_imj = flx_imj * (CUDA_UNSTR(in, coord) - CUDA_UNSTR_AT(in, n_n1_0_0)) > 0 ? 0 : flx_imj;

        double fly_ij = lap_ijp - lap_ij;
        fly_ij = fly_ij * (CUDA_UNSTR_AT(in, n_0_p1_0) - CUDA_UNSTR(in, coord)) > 0 ? 0 : fly_ij;

        double fly_ijm = lap_ij - lap_ijm;
        fly_ijm = fly_ijm * (CUDA_UNSTR(in, coord) - CUDA_UNSTR_AT(in, n_0_n1_0)) > 0 ? 0 : fly_ijm;

        CUDA_UNSTR(out, coord) =
            CUDA_UNSTR(in, coord)
            - CUDA_UNSTR(coeff, coord) * (flx_ij - flx_imj + fly_ij - fly_ijm);
        
        // for debugging purposes:
        #ifdef HDIFF_DEBUG
        CUDA_UNSTR(dbg_lap, coord) = lap_ij;
        CUDA_UNSTR_NEIGH(dbg_lap, coord, -1, 0, 0) = lap_imj;
        CUDA_UNSTR_NEIGH(dbg_lap, coord, 0, -1, 0) = lap_ijm;
        CUDA_UNSTR_NEIGH(dbg_lap, coord, +1, 0, 0) = lap_ipj;
        CUDA_UNSTR_NEIGH(dbg_lap, coord, 0, +1, 0) = lap_ijp;
        CUDA_UNSTR(dbg_flx, coord) = flx_ij;
        CUDA_UNSTR_NEIGH(dbg_flx, coord, -1, 0, 0) = flx_imj;
        CUDA_UNSTR(dbg_fly, coord) = fly_ij;
        CUDA_UNSTR_NEIGH(dbg_fly, coord, 0, -1, 0) = fly_ijm;
        #endif
 
    }

        /** Naive implementation of a unstructured grid horizontal diffusion
     * kernel. Runs index calculations in every k-iteration.
     */
     __global__
     void kernel_naive_kloop(Info info,
                             CudaUnstructuredGrid3DInfo<double> in,
                             CudaUnstructuredGrid3DInfo<double> out,
                             CudaUnstructuredGrid3DInfo<double> coeff
                             #ifdef HDIFF_DEBUG
                             , CudaUnstructuredGrid3DInfo<double> dbg_lap
                             , CudaUnstructuredGrid3DInfo<double> dbg_flx
                             , CudaUnstructuredGrid3DInfo<double> dbg_fly
                             #endif
                             ) {
        const int i = threadIdx.x + blockIdx.x*blockDim.x + info.halo.x;
        const int j = threadIdx.y + blockIdx.y*blockDim.y + info.halo.y;
        if(i-info.halo.x >= info.inner_size.x || j-info.halo.y >= info.inner_size.y) {
            return;
        }
        
        for(int k = info.halo.z; k < info.inner_size.z+info.halo.z; k++) {
            coord3 coord(i, j, k);

            int n_0_0_0       = CUDA_UNSTR_INDEX(in, coord);
            int n_0_n1_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_0_0,   0, -1, 0);
            int n_0_n2_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_n1_0,  0, -1, 0);
            int n_n1_0_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_0_0,  -1, 0, 0);
            int n_n1_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_n1_0_0,  0, -1, 0);
            int n_n2_0_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_n1_0_0, -1, 0, 0);
            //int n_n2_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_n2_0_0,  0, -1, 0);
            int n_0_p1_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_0_0,   0, +1, 0);
            int n_0_p2_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_p1_0,  0, +1, 0);
            int n_p1_0_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_0_0,  +1, 0, 0);
            int n_p1_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_p1_0_0,  0, +1, 0);
            int n_p2_0_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_p1_0_0, +1, 0, 0);
            //int n_p2_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_p2_0_0,  0, +1, 0);     
            int n_n1_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_n1_0_0,  0, +1, 0);
            int n_p1_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_p1_0_0,  0, -1, 0);

            double lap_ij = 
                4 * CUDA_UNSTR_AT(in, n_0_0_0) 
                - CUDA_UNSTR_AT(in, n_n1_0_0) - CUDA_UNSTR_AT(in, n_p1_0_0)
                - CUDA_UNSTR_AT(in, n_0_n1_0) - CUDA_UNSTR_AT(in, n_0_p1_0);
            double lap_imj = 
                4 * CUDA_UNSTR_AT(in, n_n1_0_0)
                - CUDA_UNSTR_AT(in, n_n2_0_0) - CUDA_UNSTR_AT(in, n_0_0_0)
                - CUDA_UNSTR_AT(in, n_n1_n1_0) - CUDA_UNSTR_AT(in, n_n1_p1_0);
            double lap_ipj =
                4 * CUDA_UNSTR_AT(in, n_p1_0_0)
                - CUDA_UNSTR(in, coord) - CUDA_UNSTR_AT(in, n_p2_0_0)
                - CUDA_UNSTR_AT(in, n_p1_n1_0) - CUDA_UNSTR_AT(in, n_p1_p1_0);
            double lap_ijm =
                4 * CUDA_UNSTR_AT(in, n_0_n1_0)
                - CUDA_UNSTR_AT(in, n_n1_n1_0) - CUDA_UNSTR_AT(in, n_p1_n1_0)
                - CUDA_UNSTR_AT(in, n_0_n2_0) - CUDA_UNSTR(in, coord);
            double lap_ijp =
                4 * CUDA_UNSTR_AT(in, n_0_p1_0)
                - CUDA_UNSTR_AT(in, n_n1_p1_0) - CUDA_UNSTR_AT(in, n_p1_p1_0)
                - CUDA_UNSTR(in, coord) - CUDA_UNSTR_AT(in, n_0_p2_0);

            double flx_ij = lap_ipj - lap_ij;
            flx_ij = flx_ij * (CUDA_UNSTR_AT(in, n_p1_0_0) - CUDA_UNSTR(in, coord)) > 0 ? 0 : flx_ij;

            double flx_imj = lap_ij - lap_imj;
            flx_imj = flx_imj * (CUDA_UNSTR(in, coord) - CUDA_UNSTR_AT(in, n_n1_0_0)) > 0 ? 0 : flx_imj;

            double fly_ij = lap_ijp - lap_ij;
            fly_ij = fly_ij * (CUDA_UNSTR_AT(in, n_0_p1_0) - CUDA_UNSTR(in, coord)) > 0 ? 0 : fly_ij;

            double fly_ijm = lap_ij - lap_ijm;
            fly_ijm = fly_ijm * (CUDA_UNSTR(in, coord) - CUDA_UNSTR_AT(in, n_0_n1_0)) > 0 ? 0 : fly_ijm;

            CUDA_UNSTR(out, coord) =
                CUDA_UNSTR(in, coord)
                - CUDA_UNSTR(coeff, coord) * (flx_ij - flx_imj + fly_ij - fly_ijm);
            
            // for debugging purposes:
            #ifdef HDIFF_DEBUG
            CUDA_UNSTR(dbg_lap, coord) = lap_ij;
            CUDA_UNSTR_NEIGH(dbg_lap, coord, -1, 0, 0) = lap_imj;
            CUDA_UNSTR_NEIGH(dbg_lap, coord, 0, -1, 0) = lap_ijm;
            CUDA_UNSTR_NEIGH(dbg_lap, coord, +1, 0, 0) = lap_ipj;
            CUDA_UNSTR_NEIGH(dbg_lap, coord, 0, +1, 0) = lap_ijp;
            CUDA_UNSTR(dbg_flx, coord) = flx_ij;
            CUDA_UNSTR_NEIGH(dbg_flx, coord, -1, 0, 0) = flx_imj;
            CUDA_UNSTR(dbg_fly, coord) = fly_ij;
            CUDA_UNSTR_NEIGH(dbg_fly, coord, 0, -1, 0) = fly_ijm;
            #endif
        }
 
    }

    /** This kernel makes use of the regularity of the grid in the Z-direction.
     * Instead of naively resolving the neighborship relations at each k-step,
     * The locations of the neighboring cells are calculated at one level and
     * then reused, with the constant (regular) Z-step at each k-iteration.
     */
    __global__
    void kernel_idxvars(Info info,
                        CudaUnstructuredGrid3DInfo<double> in,
                        CudaUnstructuredGrid3DInfo<double> out,
                        CudaUnstructuredGrid3DInfo<double> coeff
                        #ifdef HDIFF_DEBUG
                        , CudaUnstructuredGrid3DInfo<double> dbg_lap
                        , CudaUnstructuredGrid3DInfo<double> dbg_flx
                        , CudaUnstructuredGrid3DInfo<double> dbg_fly
                        #endif
                        ) {
        const int i = threadIdx.x + blockIdx.x*blockDim.x + info.halo.x;
        const int j = threadIdx.y + blockIdx.y*blockDim.y + info.halo.y;
        if(i-info.halo.x >= info.inner_size.x || j-info.halo.y >= info.inner_size.y) {
            return;
        }

        /** Store index offsets for the current x and y coordinate, so we do
         * not have to recalculate them in every k-iteration. Instead, with
         * each iteration, the k-stride is simply added once -- thus making
         * use of the regularity of the grid in z-direction. 
         * idx of neighbor X Y Z = n_X_Y_Z with p for positive offset and 
         * n for negative offset. */
        coord3 coord(i, j, 0);
        int n_0_0_0       = CUDA_UNSTR_INDEX(in, coord);
        int n_0_n1_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_0_0,   0, -1, 0);
        int n_0_n2_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_n1_0,  0, -1, 0);
        int n_n1_0_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_0_0,  -1, 0, 0);
        int n_n1_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_n1_0_0,  0, -1, 0);
        int n_n2_0_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_n1_0_0, -1, 0, 0);
        //int n_n2_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_n2_0_0,  0, -1, 0);
        int n_0_p1_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_0_0,   0, +1, 0);
        int n_0_p2_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_p1_0,  0, +1, 0);
        int n_p1_0_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_0_0_0,  +1, 0, 0);
        int n_p1_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_p1_0_0,  0, +1, 0);
        int n_p2_0_0      = CUDA_UNSTR_NEIGHBOR_AT(in, n_p1_0_0, +1, 0, 0);
        //int n_p2_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_p2_0_0,  0, +1, 0);     
        int n_n1_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_n1_0_0,  0, +1, 0);
        int n_p1_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(in, n_p1_0_0,  0, -1, 0);

        for (int k = info.halo.z; k < info.inner_size.z + info.halo.z; k++) {
            const coord3 coord(i, j, k);

            double lap_ij = 
                4 * CUDA_UNSTR_AT(in, n_0_0_0) 
                - CUDA_UNSTR_AT(in, n_n1_0_0) - CUDA_UNSTR_AT(in, n_p1_0_0)
                - CUDA_UNSTR_AT(in, n_0_n1_0) - CUDA_UNSTR_AT(in, n_0_p1_0);
            double lap_imj = 
                4 * CUDA_UNSTR_AT(in, n_n1_0_0)
                - CUDA_UNSTR_AT(in, n_n2_0_0) - CUDA_UNSTR_AT(in, n_0_0_0)
                - CUDA_UNSTR_AT(in, n_n1_n1_0) - CUDA_UNSTR_AT(in, n_n1_p1_0);
            double lap_ipj =
                4 * CUDA_UNSTR_AT(in, n_p1_0_0)
                - CUDA_UNSTR(in, coord) - CUDA_UNSTR_AT(in, n_p2_0_0)
                - CUDA_UNSTR_AT(in, n_p1_n1_0) - CUDA_UNSTR_AT(in, n_p1_p1_0);
            double lap_ijm =
                4 * CUDA_UNSTR_AT(in, n_0_n1_0)
                - CUDA_UNSTR_AT(in, n_n1_n1_0) - CUDA_UNSTR_AT(in, n_p1_n1_0)
                - CUDA_UNSTR_AT(in, n_0_n2_0) - CUDA_UNSTR(in, coord);
            double lap_ijp =
                4 * CUDA_UNSTR_AT(in, n_0_p1_0)
                - CUDA_UNSTR_AT(in, n_n1_p1_0) - CUDA_UNSTR_AT(in, n_p1_p1_0)
                - CUDA_UNSTR(in, coord) - CUDA_UNSTR_AT(in, n_0_p2_0);

            double flx_ij = lap_ipj - lap_ij;
            flx_ij = flx_ij * (CUDA_UNSTR_AT(in, n_p1_0_0) - CUDA_UNSTR(in, coord)) > 0 ? 0 : flx_ij;

            double flx_imj = lap_ij - lap_imj;
            flx_imj = flx_imj * (CUDA_UNSTR(in, coord) - CUDA_UNSTR_AT(in, n_n1_0_0)) > 0 ? 0 : flx_imj;

            double fly_ij = lap_ijp - lap_ij;
            fly_ij = fly_ij * (CUDA_UNSTR_AT(in, n_0_p1_0) - CUDA_UNSTR(in, coord)) > 0 ? 0 : fly_ij;

            double fly_ijm = lap_ij - lap_ijm;
            fly_ijm = fly_ijm * (CUDA_UNSTR(in, coord) - CUDA_UNSTR_AT(in, n_0_n1_0)) > 0 ? 0 : fly_ijm;

            CUDA_UNSTR(out, coord) =
                CUDA_UNSTR(in, coord)
                - CUDA_UNSTR(coeff, coord) * (flx_ij - flx_imj + fly_ij - fly_ijm);
            
            // DEBUG: Output intermediate results as well
            // Disable this for better performance
            #ifdef HDIFF_DEBUG
            CUDA_UNSTR(dbg_lap, coord) = lap_ij;
            CUDA_UNSTR_NEIGH(dbg_lap, coord, -1, 0, 0) = lap_imj;
            CUDA_UNSTR_NEIGH(dbg_lap, coord, 0, -1, 0) = lap_ijm;
            #endif
            // for debugging purposes:
            #ifdef HDIFF_DEBUG
            CUDA_UNSTR(dbg_lap, coord) = lap_ij;
            CUDA_UNSTR_NEIGH(dbg_lap, coord, -1, 0, 0) = lap_imj;
            CUDA_UNSTR_NEIGH(dbg_lap, coord, 0, -1, 0) = lap_ijm;
            CUDA_UNSTR_NEIGH(dbg_lap, coord, +1, 0, 0) = lap_ipj;
            CUDA_UNSTR_NEIGH(dbg_lap, coord, 0, +1, 0) = lap_ijp;
            CUDA_UNSTR(dbg_flx, coord) = flx_ij;
            CUDA_UNSTR_NEIGH(dbg_flx, coord, -1, 0, 0) = flx_imj;
            CUDA_UNSTR(dbg_fly, coord) = fly_ij;
            CUDA_UNSTR_NEIGH(dbg_fly, coord, 0, -1, 0) = fly_ijm;
            #endif

            n_0_0_0       += in.strides.z;
            n_0_n1_0      += in.strides.z;
            n_0_n2_0      += in.strides.z;
            n_n1_0_0      += in.strides.z;
            n_n1_n1_0     += in.strides.z;
            n_n2_0_0      += in.strides.z;
            //n_n2_n1_0     += in.strides.z;
            n_0_p1_0      += in.strides.z;
            n_0_p2_0      += in.strides.z;
            n_p1_0_0      += in.strides.z;
            n_p1_p1_0     += in.strides.z;
            n_p2_0_0      += in.strides.z;
            //n_p2_p1_0     += in.strides.z;
            n_n1_p1_0     += in.strides.z;
            n_p1_n1_0     += in.strides.z;

        }
    }

};

/** Cuda implementation of different variants of the horizontal diffusion
 * kernel, both for structured and unstructured grid variants.
 *
 * For the available variants, see the HdiffCuda::Variant enum. */
class HdiffCudaUnstrBenchmark : public HdiffBaseBenchmark {

    public:

    HdiffCudaUnstrBenchmark(coord3 size, HdiffCudaUnstr::Variant variant=HdiffCudaUnstr::UnstrNaive);
    
    HdiffCudaUnstr::Variant variant;

    virtual void run();
    virtual void setup();
    virtual void teardown();
    virtual void post();
    virtual dim3 numblocks();
    virtual dim3 numthreads();

    // Return info struct for kernels
    HdiffCudaUnstr::Info get_info();

};

// IMPLEMENTATIONS

HdiffCudaUnstrBenchmark::HdiffCudaUnstrBenchmark(coord3 size, HdiffCudaUnstr::Variant variant) :
HdiffBaseBenchmark(size) {
    if(variant == HdiffCudaUnstr::UnstrNaive) {
        this->name = "hdiff-unstr-naive";
    } else if(variant == HdiffCudaUnstr::UnstrKloop) {
        this->name = "hdiff-unstr-kloop";
    } else {
        this->name = "hdiff-unstr-idxvar";
    }
    this->error = false;
    this->variant = variant;
}

void HdiffCudaUnstrBenchmark::run() {
    auto kernel_fun = &HdiffCudaUnstr::kernel_naive;
    if(this->variant == HdiffCudaUnstr::UnstrIdxvars) {
        kernel_fun = &HdiffCudaUnstr::kernel_idxvars;
    } else if(this->variant == HdiffCudaUnstr::UnstrKloop) {
        kernel_fun = &HdiffCudaUnstr::kernel_naive_kloop;
    }
    (*kernel_fun)<<<this->numblocks(), this->numthreads()>>>(
        this->get_info(),
        (dynamic_cast<CudaUnstructuredGrid3D<double>*>(this->input))->get_gridinfo(),
        (dynamic_cast<CudaUnstructuredGrid3D<double>*>(this->output))->get_gridinfo(),
        (dynamic_cast<CudaUnstructuredGrid3D<double>*>(this->coeff))->get_gridinfo()
        #ifdef HDIFF_DEBUG
        , (dynamic_cast<CudaUnstructuredGrid3D<double>*>(this->lap))->get_gridinfo()
        , (dynamic_cast<CudaUnstructuredGrid3D<double>*>(this->flx))->get_gridinfo()
        , (dynamic_cast<CudaUnstructuredGrid3D<double>*>(this->fly))->get_gridinfo()
        #endif
    );
    if(hipDeviceSynchronize() != hipSuccess) {
        this->error = true;
    }
}

dim3 HdiffCudaUnstrBenchmark::numblocks() {
    dim3 numblocks = this->Benchmark::numblocks();
    // For the vriants that use a k-loop inside the kernel, we only need one block in the k-direction
    if(this->variant == HdiffCudaUnstr::UnstrIdxvars ||
       this->variant == HdiffCudaUnstr::UnstrKloop ) {
        numblocks = dim3(numblocks.x, numblocks.y, 1);
    }
    return numblocks;
}

dim3 HdiffCudaUnstrBenchmark::numthreads() {
    dim3 numthreads = this->Benchmark::numthreads();
    // Variants with a k-loop: only one thread in the k-direction
    if(this->variant == HdiffCudaUnstr::UnstrIdxvars ||
        this->variant == HdiffCudaUnstr::UnstrKloop ) {
        numthreads = dim3(numthreads.x, numthreads.y, 1);
    }
    return numthreads;
}

void HdiffCudaUnstrBenchmark::setup() {
    this->input = CudaUnstructuredGrid3D<double>::create_regular(this->size);
    this->output = CudaUnstructuredGrid3D<double>::create_regular(this->size);
    this->coeff = CudaUnstructuredGrid3D<double>::create_regular(this->size);
    this->lap = CudaUnstructuredGrid3D<double>::create_regular(this->size);
    this->flx = CudaUnstructuredGrid3D<double>::create_regular(this->size);
    this->fly = CudaUnstructuredGrid3D<double>::create_regular(this->size);
    this->HdiffBaseBenchmark::setup();
}

void HdiffCudaUnstrBenchmark::teardown() {
    this->input->deallocate();
    this->output->deallocate();
    this->coeff->deallocate();
    this->lap->deallocate();
    this->flx->deallocate();
    this->fly->deallocate();
    this->HdiffBaseBenchmark::teardown();
}

void HdiffCudaUnstrBenchmark::post() {
    this->Benchmark::post();
    this->HdiffBaseBenchmark::post();
}

HdiffCudaUnstr::Info HdiffCudaUnstrBenchmark::get_info() {
    return { .halo = this->halo,
             .inner_size = this->input->dimensions-2*this->halo};
}

#endif