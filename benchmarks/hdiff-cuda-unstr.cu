#include "hip/hip_runtime.h"
#ifndef HDIFF_CUDA_UNSTR_H
#define HDIFF_CUDA_UNSTR_H
#include "benchmarks/benchmark.cu"
#include "benchmarks/hdiff-cuda-base.cu"
#include "coord3.cu"
#include "grids/grid.cu"
#include "grids/cuda-regular.cu"
#include "grids/cuda-unstructured.cu"

namespace HdiffCudaUnstr {

    /** Variants of this benchmark. */
    enum Variant { naive, idxvar, idxvar_kloop, idxvar_shared };

    #define GRID_ARGS const int * __restrict__ neighborships, const int z_stride, 
    #define INDEX(x_, y_, z_) (x_) + (y_)*blockDim.x*gridDim.x + (z_)*blockDim.x*gridDim.x*blockDim.y*gridDim.y
    #define NEIGHBOR(idx, x_, y_, z_) GRID_UNSTR_NEIGHBOR(neighborships, z_stride, idx, x_, y_, z_)
    #define DOUBLE_NEIGHBOR(idx, x1, y1, z1, x2, y2, z2) NEIGHBOR(NEIGHBOR(idx, x1, y1, z1), x2, y2, z2)
    
    #include "kernels/hdiff-naive.cu"

    #undef NEIGHBOR
    #undef DOUBLE_NEIGHBOR
    #undef NEIGHBOR

    #define NEIGHBOR(idx, x, y, z) GRID_UNSTR_2D_NEIGHBOR(neighborships, z_stride, idx, x, y)
    #define NEXT_Z_NEIGHBOR(idx) (idx+z_stride)
    #define K_STEP k*z_stride

    #include "kernels/hdiff-idxvar.cu"
    #include "kernels/hdiff-idxvar-kloop.cu"
    #include "kernels/hdiff-idxvar-shared.cu"

    #undef GRID_ARGS
    #undef INDEX
    #undef NEIGHBOR
    #undef NEXT_Z_NEIGHBOR
    #undef K_STEP

};

/** Cuda implementation of different variants of the horizontal diffusion
 * kernel, both for structured and unstructured grid variants.
 *
 * For the available variants, see the HdiffCuda::Variant enum. */
template<typename value_t>
class HdiffCudaUnstrBenchmark : public HdiffCudaBaseBenchmark<value_t> {

    public:

    HdiffCudaUnstrBenchmark(coord3 size, HdiffCudaUnstr::Variant variant=HdiffCudaUnstr::idxvar);
    
    HdiffCudaUnstr::Variant variant;

    virtual void run();
    virtual void setup();
    virtual void teardown();
    virtual void post();
    virtual dim3 numblocks();
    virtual dim3 numthreads();

};

// IMPLEMENTATIONS

template<typename value_t>
HdiffCudaUnstrBenchmark<value_t>::HdiffCudaUnstrBenchmark(coord3 size, HdiffCudaUnstr::Variant variant) :
HdiffCudaBaseBenchmark<value_t>(size) {
    if(variant == HdiffCudaUnstr::naive) {
        this->name = "hdiff-unstr-naive";
    } else if(variant == HdiffCudaUnstr::idxvar) {
        this->name = "hdiff-unstr-idxvar";
    } else if(variant == HdiffCudaUnstr::idxvar_kloop) {
        this->name = "hdiff-unstr-idxvar-kloop";
    } else if(variant == HdiffCudaUnstr::idxvar_shared) {
        this->name = "hdiff-unstr-idxvar-shared";
    }
    this->error = false;
    this->variant = variant;
}

template<typename value_t>
void HdiffCudaUnstrBenchmark<value_t>::run() {
    auto kernel_fun = &HdiffCudaUnstr::hdiff_idxvar<value_t>;
    int smem = 0;
    if(this->variant == HdiffCudaUnstr::naive) {
        kernel_fun = &HdiffCudaUnstr::hdiff_naive<value_t>;
    } else if(this->variant == HdiffCudaUnstr::idxvar_kloop) {
        kernel_fun = &HdiffCudaUnstr::hdiff_idxvar_kloop<value_t>;
    } else if(this->variant == HdiffCudaUnstr::idxvar_shared) {
        kernel_fun = &HdiffCudaUnstr::hdiff_idxvar_shared<value_t>;
        dim3 numthreads = this->numthreads();
        smem = numthreads.x*numthreads.y*12*sizeof(int);
    }
    CudaUnstructuredGrid3D<value_t> *unstr_input = dynamic_cast<CudaUnstructuredGrid3D<value_t>*>(this->input);
    (*kernel_fun)<<<this->numblocks(), this->numthreads(), smem>>>(
        this->get_info(),
        unstr_input->neighborships,
        unstr_input->dimensions.x*unstr_input->dimensions.y,
        this->input->pointer(coord3(0, 0, 0)),
        this->output->pointer(coord3(0, 0, 0)),
        this->coeff->pointer(coord3(0, 0, 0))
    );
    CUDA_THROW_LAST();
    CUDA_THROW( hipDeviceSynchronize() );
}

template<typename value_t>
dim3 HdiffCudaUnstrBenchmark<value_t>::numblocks() {
    dim3 numblocks = this->Benchmark::numblocks();
    // For the vriants that use a k-loop inside the kernel, we only need one block in the k-direction
    if(this->variant == HdiffCudaUnstr::idxvar_kloop) {
        numblocks = dim3(numblocks.x, numblocks.y, 1);
    }
    return numblocks;
}

template<typename value_t>
dim3 HdiffCudaUnstrBenchmark<value_t>::numthreads() {
    dim3 numthreads = this->Benchmark::numthreads();
    // Variants with a k-loop: only one thread in the k-direction
    if(this->variant == HdiffCudaUnstr::idxvar_kloop) {
        numthreads = dim3(numthreads.x, numthreads.y, 1);
    }
    return numthreads;
}

template<typename value_t>
void HdiffCudaUnstrBenchmark<value_t>::setup() {
    this->input = CudaUnstructuredGrid3D<value_t>::create_regular(this->inner_size, this->halo);
    int *neighborships = dynamic_cast<CudaUnstructuredGrid3D<value_t> *>(this->input)->neighborships;
    this->output = new CudaUnstructuredGrid3D<value_t>(this->inner_size, this->halo);
    this->coeff = new CudaUnstructuredGrid3D<value_t>(this->inner_size, this->halo);
    this->lap = new CudaUnstructuredGrid3D<value_t>(this->inner_size, this->halo);
    this->flx = new CudaUnstructuredGrid3D<value_t>(this->inner_size, this->halo);
    this->fly = new CudaUnstructuredGrid3D<value_t>(this->inner_size, this->halo);
    if(this->variant == HdiffCudaUnstr::idxvar_shared) {
        this->input->setSmemBankSize(sizeof(int));
    }
    this->HdiffCudaBaseBenchmark<value_t>::setup();
}

template<typename value_t>
void HdiffCudaUnstrBenchmark<value_t>::teardown() {
    delete this->input;
    delete this->output;
    delete this->coeff;
    delete this->lap;
    delete this->flx;
    delete this->fly;
    this->HdiffCudaBaseBenchmark<value_t>::teardown();
}

template<typename value_t>
void HdiffCudaUnstrBenchmark<value_t>::post() {
    this->Benchmark::post();
    this->HdiffCudaBaseBenchmark<value_t>::post();
}

#endif