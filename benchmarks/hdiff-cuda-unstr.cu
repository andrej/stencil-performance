#include "hip/hip_runtime.h"
#ifndef HDIFF_CUDA_UNSTR_H
#define HDIFF_CUDA_UNSTR_H
#include "benchmarks/benchmark.cu"
#include "coord3.cu"
#include "grids/grid.cu"
#include "grids/cuda-regular.cu"
#include "grids/cuda-unstructured.cu"

namespace HdiffCudaUnstr {

    /** Variants of this benchmark. */
    enum Variant { RegNaive, RegKloop, UnstrNaive, UnstrKloop, UnstrIdxvars, UnstrSharedIdxvar };

    /** Information about this benchmark for use in the kernels. */
    struct Info {
        coord3 halo;
        coord3 max_coord;
    };

    /** Naive implementation of a unstructured grid horizontal diffusion
     * kernel. Runs index calculations in every k-iteration.
     */
     template<typename value_t>
     __global__
     void kernel_naive(Info info,
                         CudaUnstructuredGrid3DInfo<value_t> grid_info,
                         value_t* in,
                         value_t* out,
                         value_t* coeff
                         #ifdef HDIFF_DEBUG
                         , value_t* dbg_lap
                         , value_t* dbg_flx
                         , value_t* dbg_fly
                         #endif
                         ) {
        const int i = threadIdx.x + blockIdx.x*blockDim.x + info.halo.x;
        const int j = threadIdx.y + blockIdx.y*blockDim.y + info.halo.y;
        const int k = threadIdx.z + blockIdx.z*blockDim.z + info.halo.z;
        if(i >= info.max_coord.x || j >= info.max_coord.y || k >= info.max_coord.z) {
            return;
        }
        
        const int n_0_0_0       = CUDA_UNSTR_INDEX(grid_info, i, j, k);
        const int n_0_n1_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_0_0_0, 0, -1, 0); /* left */
        const int n_0_n2_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_0_n1_0, 0, -1, 0); /* 2 left */
        const int n_n1_0_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_0_0_0, -1, 0, 0); /* top */
        const int n_n1_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_n1_0_0, 0, -1, 0); /* top left */
        const int n_n2_0_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_n1_0_0, -1, 0, 0); /* 2 top */
        //const int n_n2_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_n2_0_0, 0, -1, 0);
        const int n_0_p1_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_0_0_0, 0, +1, 0); /* right */
        const int n_0_p2_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_0_p1_0, 0, +1, 0); /* 2 right */
        const int n_p1_0_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_0_0_0, +1, 0, 0); /* bottom */
        const int n_p1_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_p1_0_0, 0, +1, 0); /* bottom right */
        const int n_p2_0_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_p1_0_0, +1, 0, 0); /* 2 bottom */
        //const int n_p2_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_p2_0_0, 0, +1, 0);     
        const int n_n1_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_n1_0_0, 0, +1, 0); /* top right */
        const int n_p1_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_p1_0_0, 0, -1, 0); /* bottom left */

        const value_t lap_ij = 
            4 * in[n_0_0_0] 
            - in[n_n1_0_0] - in[n_p1_0_0]
            - in[n_0_n1_0] - in[n_0_p1_0];
        const value_t lap_imj = 
            4 * in[n_n1_0_0]
            - in[n_n2_0_0] - in[n_0_0_0]
            - in[n_n1_n1_0] - in[n_n1_p1_0];
        const value_t lap_ipj =
            4 * in[n_p1_0_0]
            - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_p2_0_0]
            - in[n_p1_n1_0] - in[n_p1_p1_0];
        const value_t lap_ijm =
            4 * in[n_0_n1_0]
            - in[n_n1_n1_0] - in[n_p1_n1_0]
            - in[n_0_n2_0] - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)];
        const value_t lap_ijp =
            4 * in[n_0_p1_0]
            - in[n_n1_p1_0] - in[n_p1_p1_0]
            - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_0_p2_0];

        value_t flx_ij = lap_ipj - lap_ij;
        flx_ij = flx_ij * (in[n_p1_0_0] - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)]) > 0 ? 0 : flx_ij;

        value_t flx_imj = lap_ij - lap_imj;
        flx_imj = flx_imj * (in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_n1_0_0]) > 0 ? 0 : flx_imj;

        value_t fly_ij = lap_ijp - lap_ij;
        fly_ij = fly_ij * (in[n_0_p1_0] - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)]) > 0 ? 0 : fly_ij;

        value_t fly_ijm = lap_ij - lap_ijm;
        fly_ijm = fly_ijm * (in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_0_n1_0]) > 0 ? 0 : fly_ijm;

        out[CUDA_UNSTR_INDEX(grid_info, i, j, k)] =
            in[CUDA_UNSTR_INDEX(grid_info, i, j, k)]
            - coeff[CUDA_UNSTR_INDEX(grid_info, i, j, k)] * (flx_ij - flx_imj + fly_ij - fly_ijm);
        
        // for debugging purposes:
        #ifdef HDIFF_DEBUG
        dbg_lap[CUDA_UNSTR_INDEX(grid_info, i, j, k)] = lap_ij;
        dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, -1, 0, 0)] = lap_imj;
        dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, 0, -1, 0)] = lap_ijm;
        dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, +1, 0, 0)] = lap_ipj;
        dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, 0, +1, 0)] = lap_ijp;
        dbg_flx[CUDA_UNSTR_INDEX(grid_info, i, j, k)] = flx_ij;
        dbg_flx[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, -1, 0, 0)] = flx_imj;
        dbg_fly[CUDA_UNSTR_INDEX(grid_info, i, j, k)] = fly_ij;
        dbg_fly[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, 0, -1, 0)] = fly_ijm;
        #endif
 
    }

    /** Naive implementation of a unstructured grid horizontal diffusion
     * kernel. Runs index calculations in every k-iteration.
     */
     template<typename value_t>
     __global__
     void kernel_naive_kloop(Info info,
                             CudaUnstructuredGrid3DInfo<value_t> grid_info,
                             value_t* in,
                             value_t* out,
                             value_t* coeff
                             #ifdef HDIFF_DEBUG
                             , value_t* dbg_lap
                             , value_t* dbg_flx
                             , value_t* dbg_fly
                             #endif
                             ) {
        const int i = threadIdx.x + blockIdx.x*blockDim.x + info.halo.x;
        const int j = threadIdx.y + blockIdx.y*blockDim.y + info.halo.y;
        if(i >= info.max_coord.x || j >= info.max_coord.y) {
            return;
        }
        
        for(int k = info.halo.z; k < info.max_coord.z; k++) {

            int n_0_0_0       = CUDA_UNSTR_INDEX(grid_info, i, j, k);
            int n_0_n1_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_0_0_0, 0, -1, 0);
            int n_0_n2_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_0_n1_0, 0, -1, 0);
            int n_n1_0_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_0_0_0, -1, 0, 0);
            int n_n1_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_n1_0_0, 0, -1, 0);
            int n_n2_0_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_n1_0_0, -1, 0, 0);
            //int n_n2_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_n2_0_0, 0, -1, 0);
            int n_0_p1_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_0_0_0, 0, +1, 0);
            int n_0_p2_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_0_p1_0, 0, +1, 0);
            int n_p1_0_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_0_0_0, +1, 0, 0);
            int n_p1_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_p1_0_0, 0, +1, 0);
            int n_p2_0_0      = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_p1_0_0, +1, 0, 0);
            //int n_p2_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_p2_0_0, 0, +1, 0);     
            int n_n1_p1_0     = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_n1_0_0, 0, +1, 0);
            int n_p1_n1_0     = CUDA_UNSTR_NEIGHBOR_AT(grid_info, n_p1_0_0, 0, -1, 0);

            value_t lap_ij = 
                4 * in[n_0_0_0] 
                - in[n_n1_0_0] - in[n_p1_0_0]
                - in[n_0_n1_0] - in[n_0_p1_0];
            value_t lap_imj = 
                4 * in[n_n1_0_0]
                - in[n_n2_0_0] - in[n_0_0_0]
                - in[n_n1_n1_0] - in[n_n1_p1_0];
            value_t lap_ipj =
                4 * in[n_p1_0_0]
                - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_p2_0_0]
                - in[n_p1_n1_0] - in[n_p1_p1_0];
            value_t lap_ijm =
                4 * in[n_0_n1_0]
                - in[n_n1_n1_0] - in[n_p1_n1_0]
                - in[n_0_n2_0] - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)];
            value_t lap_ijp =
                4 * in[n_0_p1_0]
                - in[n_n1_p1_0] - in[n_p1_p1_0]
                - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_0_p2_0];

            value_t flx_ij = lap_ipj - lap_ij;
            flx_ij = flx_ij * (in[n_p1_0_0] - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)]) > 0 ? 0 : flx_ij;

            value_t flx_imj = lap_ij - lap_imj;
            flx_imj = flx_imj * (in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_n1_0_0]) > 0 ? 0 : flx_imj;

            value_t fly_ij = lap_ijp - lap_ij;
            fly_ij = fly_ij * (in[n_0_p1_0] - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)]) > 0 ? 0 : fly_ij;

            value_t fly_ijm = lap_ij - lap_ijm;
            fly_ijm = fly_ijm * (in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_0_n1_0]) > 0 ? 0 : fly_ijm;

            out[CUDA_UNSTR_INDEX(grid_info, i, j, k)] =
                in[CUDA_UNSTR_INDEX(grid_info, i, j, k)]
                - coeff[CUDA_UNSTR_INDEX(grid_info, i, j, k)] * (flx_ij - flx_imj + fly_ij - fly_ijm);
            
            // for debugging purposes:
            #ifdef HDIFF_DEBUG
            dbg_lap[CUDA_UNSTR_INDEX(grid_info, i, j, k)] = lap_ij;
            dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, -1, 0, 0)] = lap_imj;
            dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, 0, -1, 0)] = lap_ijm;
            dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, +1, 0, 0)] = lap_ipj;
            dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, 0, +1, 0)] = lap_ijp;
            dbg_flx[CUDA_UNSTR_INDEX(grid_info, i, j, k)] = flx_ij;
            dbg_flx[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, -1, 0, 0)] = flx_imj;
            dbg_fly[CUDA_UNSTR_INDEX(grid_info, i, j, k)] = fly_ij;
            dbg_fly[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, 0, -1, 0)] = fly_ijm;
            #endif
        }
 
    }

    /** This kernel makes use of the regularity of the grid in the Z-direction.
     * Instead of naively resolving the neighborship relations at each k-step,
     * The locations of the neighboring cells are calculated at one level and
     * then reused, with the constant (regular) Z-step at each k-iteration.
     */
    template<typename value_t>
    __global__
    void kernel_idxvars(Info info,
                        CudaUnstructuredGrid3DInfo<value_t> grid_info,
                        value_t* in,
                        value_t* out,
                        value_t* coeff
                        #ifdef HDIFF_DEBUG
                        , value_t* dbg_lap
                        , value_t* dbg_flx
                        , value_t* dbg_fly
                        #endif
                        ) {
        const int i = threadIdx.x + blockIdx.x*blockDim.x + info.halo.x;
        const int j = threadIdx.y + blockIdx.y*blockDim.y + info.halo.y;
        if(i >= info.max_coord.x || j >= info.max_coord.y) {
            return;
        }

        /** Store index offsets for the current x and y coordinate, so we do
         * not have to recalculate them in every k-iteration. Instead, with
         * each iteration, the k-stride is simply added once -- thus making
         * use of the regularity of the grid in z-direction. 
         * idx of neighbor X Y Z = n_X_Y_Z with p for positive offset and 
         * n for negative offset. */
        int n_0_0_0       = CUDA_UNSTR_INDEX(grid_info, i, j, 0);
        int n_0_n1_0      = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_0_0_0, 0, -1);
        int n_0_n2_0      = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_0_n1_0, 0, -1);
        int n_n1_0_0      = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_0_0_0, -1, 0);
        int n_n1_n1_0     = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_n1_0_0, 0, -1);
        int n_n2_0_0      = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_n1_0_0, -1, 0);
        //int n_n2_n1_0     = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_n2_0_0, 0, -1);
        int n_0_p1_0      = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_0_0_0, 0, +1);
        int n_0_p2_0      = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_0_p1_0, 0, +1);
        int n_p1_0_0      = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_0_0_0, +1, 0);
        int n_p1_p1_0     = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_p1_0_0, 0, +1);
        int n_p2_0_0      = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_p1_0_0, +1, 0);
        //int n_p2_p1_0     = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_p2_0_0, 0, +1);     
        int n_n1_p1_0     = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_n1_0_0, 0, +1);
        int n_p1_n1_0     = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, n_p1_0_0, 0, -1);

        for (int k = info.halo.z; k < info.max_coord.z; k++) {

            value_t lap_ij = 
                4 * in[n_0_0_0] 
                - in[n_n1_0_0] - in[n_p1_0_0]
                - in[n_0_n1_0] - in[n_0_p1_0];
            value_t lap_imj = 
                4 * in[n_n1_0_0]
                - in[n_n2_0_0] - in[n_0_0_0]
                - in[n_n1_n1_0] - in[n_n1_p1_0];
            value_t lap_ipj =
                4 * in[n_p1_0_0]
                - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_p2_0_0]
                - in[n_p1_n1_0] - in[n_p1_p1_0];
            value_t lap_ijm =
                4 * in[n_0_n1_0]
                - in[n_n1_n1_0] - in[n_p1_n1_0]
                - in[n_0_n2_0] - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)];
            value_t lap_ijp =
                4 * in[n_0_p1_0]
                - in[n_n1_p1_0] - in[n_p1_p1_0]
                - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_0_p2_0];

            value_t flx_ij = lap_ipj - lap_ij;
            flx_ij = flx_ij * (in[n_p1_0_0] - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)]) > 0 ? 0 : flx_ij;

            value_t flx_imj = lap_ij - lap_imj;
            flx_imj = flx_imj * (in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_n1_0_0]) > 0 ? 0 : flx_imj;

            value_t fly_ij = lap_ijp - lap_ij;
            fly_ij = fly_ij * (in[n_0_p1_0] - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)]) > 0 ? 0 : fly_ij;

            value_t fly_ijm = lap_ij - lap_ijm;
            fly_ijm = fly_ijm * (in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_0_n1_0]) > 0 ? 0 : fly_ijm;

            out[CUDA_UNSTR_INDEX(grid_info, i, j, k)] =
                in[CUDA_UNSTR_INDEX(grid_info, i, j, k)]
                - coeff[CUDA_UNSTR_INDEX(grid_info, i, j, k)] * (flx_ij - flx_imj + fly_ij - fly_ijm);
            
            // DEBUG: Output intermediate results as well
            // Disable this for better performance
            #ifdef HDIFF_DEBUG
            dbg_lap[CUDA_UNSTR_INDEX(grid_info, i, j, k)] = lap_ij;
            dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, -1, 0, 0)] = lap_imj;
            dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, 0, -1, 0)] = lap_ijm;
            dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, +1, 0, 0)] = lap_ipj;
            dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, 0, +1, 0)] = lap_ijp;
            dbg_flx[CUDA_UNSTR_INDEX(grid_info, i, j, k)] = flx_ij;
            dbg_flx[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, -1, 0, 0)] = flx_imj;
            dbg_fly[CUDA_UNSTR_INDEX(grid_info, i, j, k)] = fly_ij;
            dbg_fly[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, 0, -1, 0)] = fly_ijm;
            #endif

            // Make use of regularity in Z-direciton: neighbors are exactly the
            // same, just one Z-stride apart.
            n_0_0_0       += grid_info.strides.z;
            n_0_n1_0      += grid_info.strides.z;
            n_0_n2_0      += grid_info.strides.z;
            n_n1_0_0      += grid_info.strides.z;
            n_n1_n1_0     += grid_info.strides.z;
            n_n2_0_0      += grid_info.strides.z;
            //n_n2_n1_0     += grid_info.strides.z;
            n_0_p1_0      += grid_info.strides.z;
            n_0_p2_0      += grid_info.strides.z;
            n_p1_0_0      += grid_info.strides.z;
            n_p1_p1_0     += grid_info.strides.z;
            n_p2_0_0      += grid_info.strides.z;
            //n_p2_p1_0     += grid_info.strides.z;
            n_n1_p1_0     += grid_info.strides.z;
            n_p1_n1_0     += grid_info.strides.z;

        }
    }

    /** A designated kernel invocation (at k=0) loads the neighborship relation
     * for the given x and y coordinates of this kernel. The other kernel
     * invocations at higher levels rely on shared memory to access the
     * neighborship information.
     */
     template<typename value_t>
     __global__
     void kernel_shared_idxvars(Info info,
                                CudaUnstructuredGrid3DInfo<value_t> grid_info,
                                value_t* in,
                                value_t* out,
                                value_t* coeff
                                #ifdef HDIFF_DEBUG
                                , value_t* dbg_lap
                                , value_t* dbg_flx
                                , value_t* dbg_fly
                                #endif
                                ) {
        const int i = threadIdx.x + blockIdx.x*blockDim.x + info.halo.x;
        const int j = threadIdx.y + blockIdx.y*blockDim.y + info.halo.y;
        const int k = threadIdx.z + blockIdx.z*blockDim.z + info.halo.z;
        if(i >= info.max_coord.x || j >= info.max_coord.y || k >= info.max_coord.z) {
            return;
        }
        
        extern __shared__ int smem[]; // stores four neighbors of cell i at smem[i*4]
        const int local_idx = (threadIdx.x + threadIdx.y*blockDim.x) * 12;
        const int global_idx_2d = CUDA_UNSTR_INDEX(grid_info, i, j, 0);

        if(k % blockDim.z == 0) {
            // We are the thread responsible for looking up neighbor info
            /*  0 -1 */ smem[local_idx+0] = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, global_idx_2d, 0, -1);
            /*  0 -2 */ smem[local_idx+1] = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, smem[local_idx+0], 0, -1);
            /* -1  0 */ smem[local_idx+2] = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, global_idx_2d, -1, 0);
            /* -1 -1 */ smem[local_idx+3] = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, smem[local_idx+2], 0, -1);
            /* -2  0 */ smem[local_idx+4] = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, smem[local_idx+2], -1, 0);
            //n_n2_n1_0     = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, smem_n_n2_0_0, 0, -1);
            /*  0 +1 */ smem[local_idx+5] = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, global_idx_2d, 0, +1);
            /*  0 +2 */ smem[local_idx+6] = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, smem[local_idx+5], 0, +1);
            /* +1  0 */ smem[local_idx+7] = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, global_idx_2d, +1, 0);
            /* +1 +1 */ smem[local_idx+8] = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, smem[local_idx+7], 0, +1);
            /* +2  0 */ smem[local_idx+9] = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, smem[local_idx+7], +1, 0);
            //n_p2_p1_0     = CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, smem_n_p2_0_0, 0, +1);     
            /* -1 +1 */ smem[local_idx+10]= CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, smem[local_idx+2], 0, +1);
            /* +1 -1 */ smem[local_idx+11]= CUDA_UNSTR_NEIGHBOR_AT_UNSAFE(grid_info, smem[local_idx+7], 0, -1);
        }
        
        __syncthreads();
        const int k_step = k*grid_info.strides.z;
        const int n_0_0_0       = global_idx_2d + k_step;
        const int n_0_n1_0      = smem[local_idx+0] + k_step;
        const int n_0_n2_0      = smem[local_idx+1] + k_step;
        const int n_n1_0_0      = smem[local_idx+2] + k_step;
        const int n_n1_n1_0     = smem[local_idx+3] + k_step;
        const int n_n2_0_0      = smem[local_idx+4] + k_step;
        //const int n_n2_n1_0     = smem_n_n2_n1_0 + k_step;
        const int n_0_p1_0      = smem[local_idx+5] + k_step;
        const int n_0_p2_0      = smem[local_idx+6] + k_step;
        const int n_p1_0_0      = smem[local_idx+7] + k_step;
        const int n_p1_p1_0     = smem[local_idx+8] + k_step;
        const int n_p2_0_0      = smem[local_idx+9] + k_step;
        //const int n_p2_p1_0     = smem_n_p2_p1_0 + k_step;
        const int n_n1_p1_0     = smem[local_idx+10] + k_step;
        const int n_p1_n1_0     = smem[local_idx+11] + k_step;

        const value_t lap_ij = 
            4 * in[n_0_0_0] 
            - in[n_n1_0_0] - in[n_p1_0_0]
            - in[n_0_n1_0] - in[n_0_p1_0];
        const value_t lap_imj = 
            4 * in[n_n1_0_0]
            - in[n_n2_0_0] - in[n_0_0_0]
            - in[n_n1_n1_0] - in[n_n1_p1_0];
        const value_t lap_ipj =
            4 * in[n_p1_0_0]
            - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_p2_0_0]
            - in[n_p1_n1_0] - in[n_p1_p1_0];
        const value_t lap_ijm =
            4 * in[n_0_n1_0]
            - in[n_n1_n1_0] - in[n_p1_n1_0]
            - in[n_0_n2_0] - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)];
        const value_t lap_ijp =
            4 * in[n_0_p1_0]
            - in[n_n1_p1_0] - in[n_p1_p1_0]
            - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_0_p2_0];

        value_t flx_ij = lap_ipj - lap_ij;
        flx_ij = flx_ij * (in[n_p1_0_0] - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)]) > 0 ? 0 : flx_ij;

        value_t flx_imj = lap_ij - lap_imj;
        flx_imj = flx_imj * (in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_n1_0_0]) > 0 ? 0 : flx_imj;

        value_t fly_ij = lap_ijp - lap_ij;
        fly_ij = fly_ij * (in[n_0_p1_0] - in[CUDA_UNSTR_INDEX(grid_info, i, j, k)]) > 0 ? 0 : fly_ij;

        value_t fly_ijm = lap_ij - lap_ijm;
        fly_ijm = fly_ijm * (in[CUDA_UNSTR_INDEX(grid_info, i, j, k)] - in[n_0_n1_0]) > 0 ? 0 : fly_ijm;

        out[CUDA_UNSTR_INDEX(grid_info, i, j, k)] =
            in[CUDA_UNSTR_INDEX(grid_info, i, j, k)]
            - coeff[CUDA_UNSTR_INDEX(grid_info, i, j, k)] * (flx_ij - flx_imj + fly_ij - fly_ijm);
        
        // for debugging purposes:
        #ifdef HDIFF_DEBUG
        dbg_lap[CUDA_UNSTR_INDEX(grid_info, i, j, k)] = lap_ij;
        dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, -1, 0, 0)] = lap_imj;
        dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, 0, -1, 0)] = lap_ijm;
        dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, +1, 0, 0)] = lap_ipj;
        dbg_lap[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, 0, +1, 0)] = lap_ijp;
        dbg_flx[CUDA_UNSTR_INDEX(grid_info, i, j, k)] = flx_ij;
        dbg_flx[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, -1, 0, 0)] = flx_imj;
        dbg_fly[CUDA_UNSTR_INDEX(grid_info, i, j, k)] = fly_ij;
        dbg_fly[CUDA_UNSTR_NEIGHBOR(grid_info, i, j, k, 0, -1, 0)] = fly_ijm;
        #endif
 
    }


};

/** Cuda implementation of different variants of the horizontal diffusion
 * kernel, both for structured and unstructured grid variants.
 *
 * For the available variants, see the HdiffCuda::Variant enum. */
template<typename value_t>
class HdiffCudaUnstrBenchmark : public HdiffBaseBenchmark<value_t> {

    public:

    HdiffCudaUnstrBenchmark(coord3 size, HdiffCudaUnstr::Variant variant=HdiffCudaUnstr::UnstrNaive);
    
    HdiffCudaUnstr::Variant variant;

    virtual void run();
    virtual void setup();
    virtual void teardown();
    virtual void post();
    virtual dim3 numblocks();
    virtual dim3 numthreads();

    // Return info struct for kernels
    HdiffCudaUnstr::Info get_info();

};

// IMPLEMENTATIONS

template<typename value_t>
HdiffCudaUnstrBenchmark<value_t>::HdiffCudaUnstrBenchmark(coord3 size, HdiffCudaUnstr::Variant variant) :
HdiffBaseBenchmark<value_t>(size) {
    if(variant == HdiffCudaUnstr::UnstrNaive) {
        this->name = "hdiff-unstr-naive";
    } else if(variant == HdiffCudaUnstr::UnstrKloop) {
        this->name = "hdiff-unstr-kloop";
    } else if(variant == HdiffCudaUnstr::UnstrSharedIdxvar) {
        this->name = "hdiff-unstr-shared";
    } else {
        this->name = "hdiff-unstr-idxvar";
    }
    this->error = false;
    this->variant = variant;
}

template<typename value_t>
void HdiffCudaUnstrBenchmark<value_t>::run() {
    auto kernel_fun = &HdiffCudaUnstr::kernel_naive<value_t>;
    int smem = 0;
    if(this->variant == HdiffCudaUnstr::UnstrIdxvars) {
        kernel_fun = &HdiffCudaUnstr::kernel_idxvars<value_t>;
    } else if(this->variant == HdiffCudaUnstr::UnstrKloop) {
        kernel_fun = &HdiffCudaUnstr::kernel_naive_kloop<value_t>;
    } else if(this->variant == HdiffCudaUnstr::UnstrSharedIdxvar) {
        kernel_fun = &HdiffCudaUnstr::kernel_shared_idxvars<value_t>;
        dim3 numthreads = this->numthreads();
        smem = numthreads.x*numthreads.y*12*sizeof(int);
    }
    (*kernel_fun)<<<this->numblocks(), this->numthreads(), smem>>>(
        this->get_info(),
        (dynamic_cast<CudaUnstructuredGrid3D<value_t>*>(this->input))->get_gridinfo(),
        this->input->data,
        this->output->data,
        this->coeff->data
        #ifdef HDIFF_DEBUG
        , this->lap->data
        , this->flx->data
        , this->fly->data
        #endif
    );
    if(hipDeviceSynchronize() != hipSuccess) {
        this->error = true;
    }
}

template<typename value_t>
dim3 HdiffCudaUnstrBenchmark<value_t>::numblocks() {
    dim3 numblocks = this->Benchmark::numblocks();
    // For the vriants that use a k-loop inside the kernel, we only need one block in the k-direction
    if(this->variant == HdiffCudaUnstr::UnstrIdxvars ||
       this->variant == HdiffCudaUnstr::UnstrKloop ) {
        numblocks = dim3(numblocks.x, numblocks.y, 1);
    }
    return numblocks;
}

template<typename value_t>
dim3 HdiffCudaUnstrBenchmark<value_t>::numthreads() {
    dim3 numthreads = this->Benchmark::numthreads();
    // Variants with a k-loop: only one thread in the k-direction
    if(this->variant == HdiffCudaUnstr::UnstrIdxvars ||
        this->variant == HdiffCudaUnstr::UnstrKloop ) {
        numthreads = dim3(numthreads.x, numthreads.y, 1);
    }
    return numthreads;
}

template<typename value_t>
void HdiffCudaUnstrBenchmark<value_t>::setup() {
    this->input = CudaUnstructuredGrid3D<value_t>::create_regular(this->size);
    //this->output = CudaUnstructuredGrid3D<value_t>::create_regular(this->size);
    //this->coeff = CudaUnstructuredGrid3D<value_t>::create_regular(this->size);
    //this->lap = CudaUnstructuredGrid3D<value_t>::create_regular(this->size);
    //this->flx = CudaUnstructuredGrid3D<value_t>::create_regular(this->size);
    //this->fly = CudaUnstructuredGrid3D<value_t>::create_regular(this->size);
    int *neighbor_data = dynamic_cast<CudaUnstructuredGrid3D<value_t> *>(this->input)->neighbor_data;
    this->output = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->coeff = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->lap = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->flx = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->fly = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->HdiffBaseBenchmark<value_t>::setup();
    int s1 = hipMemPrefetchAsync(this->input->data, this->input->size, 0);
    int s2 = hipMemPrefetchAsync(this->output->data, this->output->size, 0);
    int s3 = hipMemPrefetchAsync(this->coeff->data, this->coeff->size, 0);
    #ifdef HDIFF_DEBUG
    int s4 = hipMemPrefetchAsync(this->lap->data, this->lap->size, 0);
    int s5 = hipMemPrefetchAsync(this->flx->data, this->flx->size, 0);
    int s6 = hipMemPrefetchAsync(this->fly->data, this->fly->size, 0);
    #endif
    if( s1 != hipSuccess || s2 != hipSuccess || s3 != hipSuccess
        #ifdef HDIFF_DEBUG
            || s4 != hipSuccess || s5 != hipSuccess || s6 != hipSuccess
        #endif
    ) {
        throw std::runtime_error("unable to prefetch memory");
    }
}

template<typename value_t>
void HdiffCudaUnstrBenchmark<value_t>::teardown() {
    this->input->deallocate();
    this->output->deallocate();
    this->coeff->deallocate();
    this->lap->deallocate();
    this->flx->deallocate();
    this->fly->deallocate();
    delete this->input;
    delete this->output;
    delete this->coeff;
    delete this->lap;
    delete this->flx;
    delete this->fly;
    this->HdiffBaseBenchmark<value_t>::teardown();
}

template<typename value_t>
void HdiffCudaUnstrBenchmark<value_t>::post() {
    this->Benchmark::post();
    this->HdiffBaseBenchmark<value_t>::post();
}

template<typename value_t>
HdiffCudaUnstr::Info HdiffCudaUnstrBenchmark<value_t>::get_info() {
    return { .halo = this->halo,
             .max_coord = this->input->dimensions - this->halo};
}

#endif