#ifndef FASTWAVES_UNSTR_H
#define FASTWAVES_UNSTR_H
#include "benchmarks/fastwaves-base.cu"
#include "grids/cuda-unstructured.cu"

namespace FastWavesUnstrBenchmarkNamespace {

    enum Variant { unfused, naive, idxvar, idxvar_kloop };

    #define GRID_ARGS const int *neighbor_data, const int y_stride, const int z_stride, 
    #define INDEX(x, y, z) GRID_UNSTR_INDEX(y_stride, z_stride, x, y, z)
    #define NEIGHBOR(x, y, z, x_, y_, z_) GRID_UNSTR_NEIGHBOR(neighbor_data, y_stride, z_stride, x, y, z, x_, y_, z_)
    #define NEIGHBOR_OF_INDEX(idx, x, y, z) GRID_UNSTR_NEIGHBOR_OF_INDEX(neighbor_data, z_stride, idx, x, y, z)
    #define NEXT_Z_NEIGHBOR_OF_INDEX(idx) (idx+z_stride)
    #define K_STEP k*z_stride

    #include "kernels/fastwaves-unfused.cu"

    #undef GRID_ARGS
    #undef INDEX
    #undef NEIGHBOR
    #undef NEIGHBOR_OF_INDEX
    #undef NEXT_Z_NEIGHBOR_OF_INDEX
    #undef K_STEP

}

template<typename value_t>
class FastWavesUnstrBenchmark : public FastWavesBaseBenchmark<value_t> {

    public:
    
    FastWavesUnstrBenchmark(coord3 size, FastWavesUnstrBenchmarkNamespace::Variant variant);
    
    FastWavesUnstrBenchmarkNamespace::Variant variant;

    void setup();
    void run();
    //dim3 numthreads();
    //dim3 numblocks();

};

template<typename value_t>
FastWavesUnstrBenchmark<value_t>::FastWavesUnstrBenchmark(coord3 size, FastWavesUnstrBenchmarkNamespace::Variant variant) :
FastWavesBaseBenchmark<value_t>(size),
variant(variant) {
    if(this->variant == FastWavesUnstrBenchmarkNamespace::unfused) {
        this->name = "fastwaves-unstr-unfused";
    }
}

template<typename value_t>
void FastWavesUnstrBenchmark<value_t>::setup() {
    this->u_in = CudaUnstructuredGrid3D<value_t>::create_regular(this->size);
    int *neighbor_data = (dynamic_cast<CudaUnstructuredGrid3D<value_t> *>(this->u_in))->neighbor_data;
    this->v_in = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->u_tens = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->v_tens = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->rho = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->ppuv = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->fx = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->wgtfac = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->hhl = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->u_out = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    this->v_out = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    if(this->variant == FastWavesUnstrBenchmarkNamespace::unfused) {
        this->ppgk = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
        this->ppgc = new CudaUnstructuredGrid3D<value_t>(this->size, neighbor_data);
    }
    this->FastWavesBaseBenchmark<value_t>::setup(); // set initial values
}

template<typename value_t>
void FastWavesUnstrBenchmark<value_t>::run() {
    dim3 blocks = this->numblocks();
    dim3 threads = this->numthreads();
    CudaUnstructuredGrid3D<value_t> *unstr_u_in = (dynamic_cast<CudaUnstructuredGrid3D<value_t>*>(this->u_in));
    coord3 strides = coord3(1, unstr_u_in->dimensions.x, unstr_u_in->dimensions.x*unstr_u_in->dimensions.y);
    int *neighbor_data = unstr_u_in->neighbor_data;
    // Unfused: Call kernels one by one
    if(this->variant == FastWavesUnstrBenchmarkNamespace::unfused) {
        FastWavesUnstrBenchmarkNamespace::fastwaves_ppgk<<<blocks, threads>>>(
            this->get_info(),
            this->c_flat_limit,
            neighbor_data, strides.y, strides.z,
            this->ppuv->data,
            this->wgtfac->data,
            this->ppgk->data);
        FastWavesUnstrBenchmarkNamespace::fastwaves_ppgc<<<blocks, threads>>>(
            this->get_info(),
            neighbor_data, strides.y, strides.z,
            this->c_flat_limit,
            this->ppgk->data,
            this->ppgc->data);
        FastWavesUnstrBenchmarkNamespace::fastwaves_ppgrad_uv<<<blocks, threads>>>(
            this->get_info(),
            neighbor_data, strides.y, strides.z,
            this->ppuv->data,
            this->ppgc->data,
            this->hhl->data,
            this->v_in->data,
            this->u_in->data,
            this->v_tens->data,
            this->u_tens->data,
            this->rho->data,
            this->fx->data,
            this->edadlat,
            this->dt_small,
            this->c_flat_limit,
            this->u_out->data,
            this->v_out->data);
        hipDeviceSynchronize();
    }
}

#endif