#include "hip/hip_runtime.h"
#ifndef CUDA_BENCHMARK_H
#define CUDA_BENCHMARK_H
#include <float.h>
#include <string>
#include <cmath>
#include <chrono>
#include <stdexcept>
#include <sstream>
#include <algorithm>
#include <vector>
#include <numeric>
#ifdef CUDA_PROFILER
#include <hip/hip_runtime_api.h>
#endif
#include "coord3.cu"
#include "grids/coord3-base.cu"

using clk = std::chrono::high_resolution_clock;

/** The maxnumthreads limits are only enforced if the total nubmer of threads
 * (product of x, y and z) is exceeded. It is therefore well possible to have
 * more threads in a given dimension, provided that the other dimensions are
 * accordingly smaller. Note that it leads to errors to try and launch a cuda
 * kernel with too many threads. */
#ifndef CUDA_MAXNUMTHREADS_X
#define CUDA_MAXNUMTHREADS_X 16
#define CUDA_MAXNUMTHREADS_Y 16
#define CUDA_MAXNUMTHREADS_Z 4
#endif

/** Benchmark result: Average, minimum and maximum runtime in seconds. */
typedef struct { 
	struct {double avg; double median; double min; double max;} runtime;
	bool error; 
} benchmark_result_t;

/** Benchmark operating on some three dimensional grid
 *
 * This is mainly a wrapper around a run() function which the subclass should
 * overwrite.
 *
 * The value of output_grid() can be verified for correctness against another
 * grid by calling verify()
 *
 * Subclasses should overwrite the run method, which is supposed to operate on
 * the input Grid.
 */
class Benchmark {

	public:

	Benchmark();
	Benchmark(coord3 size);

	coord3 size;
    dim3 _numblocks;
    dim3 _numthreads;
	std::string name;
	bool error = false;
    benchmark_result_t results;
    bool quiet = true;
    /** Turn verification off if you are sure the benchmark computes the
     * correct result and you do not want to waste time computing the
     * the reference. */
    bool do_verify = true;
    int runs = 1;

    /** Pointers to command-line arguments specific to this benchmark.
     * In main, all arguments that follow a benchmark name are considered
     * specific to that benchmark, and the pointer to the first one of those
     * is passed in argv. In argc, we have the number of arguments until the 
     * next benchmark name or the end of the command. */
    int argc;
    char **argv;
    virtual void parse_args(); /**< do some setup based on argc, argv */

	/** Subclasses (benchmarks) must at least overwrite this function an perform
	 * the computations to be benchmarked inside here. */
	virtual void run() = 0;

	/** Executes a certain number of runs of the given benchmark and stores some
	 * metrics in this->results. */
	benchmark_result_t execute();
	
	/** Compares the value in each cell of this->output grid with the given
	 * reference grid and returns true only if all the cells match (up to the
     * optionally given tolerance). */
    template<typename value_t>
	/*virtual*/ bool verify(Grid<value_t, coord3> *reference, Grid<value_t, coord3> *other, double tol=1e-5);

	// Setup and teardown are called when the benchmark is initialized, only once
	virtual void setup() {};
	virtual void teardown() {};

	// Pre and post are called for each iteration of the benchmark, i.e. once per run
	virtual void pre() {};
	virtual void post();

	// Cuda specific: number of threads and blocks to execute the benchmark in
	// May be used by the benchmark implementation in run() to determine how many
	// threads and blocks to launch the kernel in
	virtual dim3 numthreads();
	virtual dim3 numblocks();

};

/** Computes the median of a vector of (unsorted) values. */
template<typename T>
T median(std::vector<T> vec);

// IMPLEMENTATIONS

template<typename T>
T median<T>(std::vector<T> vec) {
    if(vec.size() % 2 == 0) {
        std::nth_element(vec.begin(), vec.begin()+vec.size()/2+1, vec.end());
        return (vec[vec.size()/2]+vec[vec.size()/2+1])/2;
    } else {
        std::nth_element(vec.begin(), vec.begin()+vec.size()/2, vec.end());
        return vec[vec.size()/2];
    }
}

Benchmark::Benchmark() {}

Benchmark::Benchmark(coord3 size) : size(size) {}

void Benchmark::post() {
    if(hipGetLastError() != hipSuccess) {
        this->error = true;
        std::ostringstream msg;
        dim3 nblocks = this->numblocks();
        dim3 nthreads = this->numthreads();
        msg << "Unable to run kernel with (" << nblocks.x << ", " << nblocks.y << ", " << nblocks.z << 
               ") blocks and (" << nthreads.x << ", " << nthreads.y << ", " << nthreads.z << ") threads.";
        throw std::runtime_error(msg.str());
    }
}

dim3 Benchmark::numblocks() {
    if(this->_numblocks.x != 0 &&
        this->_numblocks.y != 0 &&
        this->_numblocks.z != 0) {
        return this->_numblocks;
    }
    dim3 numthreads = this->numthreads();
    int x = (this->size.x + numthreads.x - 1) / numthreads.x;
    int y = (this->size.y + numthreads.y - 1) / numthreads.y;
    int z = (this->size.z + numthreads.z - 1) / numthreads.z;
    return dim3( (unsigned int) x, (unsigned int) y, (unsigned int) z );
}

dim3 Benchmark::numthreads() {
    if(this->_numthreads.x != 0 &&
        this->_numthreads.y != 0 &&
        this->_numthreads.z != 0) {
        return this->_numthreads;
    }
    int x = (this->size.x + this->_numblocks.x - 1) / this->_numblocks.x;
    int y = (this->size.y + this->_numblocks.y - 1) / this->_numblocks.y;
    int z = (this->size.z + this->_numblocks.z - 1) / this->_numblocks.z;
    if (x*y*z > CUDA_MAXNUMTHREADS_X*CUDA_MAXNUMTHREADS_Y*CUDA_MAXNUMTHREADS_Z) {
        // The limiting is only done if the total maximum is exceeded
        x = std::min(x, CUDA_MAXNUMTHREADS_X);
        y = std::min(y, CUDA_MAXNUMTHREADS_Y);
        z = std::min(z, CUDA_MAXNUMTHREADS_Z);
    }
    return dim3( (unsigned int) x, (unsigned int) y, (unsigned int) z );
}

benchmark_result_t Benchmark::execute() {
	this->setup();
    bool error = false;
    std::vector<double> runtimes;
    for(int i=-1; i<this->runs; i++) {
        this->pre();
        #ifdef CUDA_PROFILER
        hipProfilerStart();
        #endif

        clk::time_point start = clk::now();
        this->run();
        clk::time_point stop = clk::now();

        #ifdef CUDA_PROFILER
        hipProfilerStop();
        #endif
        this->post();
        error = error || this->error;
        if(i == -1) {
            // First run is untimed, as Cuda recompiles the kernel on first run which would distort our measurements.
            continue;
        }
        double runtime = std::chrono::duration_cast<std::chrono::microseconds>(stop-start).count();
        runtimes.push_back(runtime);
    }
    this->teardown();
    double avg = std::accumulate(runtimes.begin(), runtimes.end(), 0.0) / runtimes.size();
    double med = median<double>(runtimes);
    double min = *std::min_element(runtimes.begin(), runtimes.end());
    double max = *std::max_element(runtimes.begin(), runtimes.end());
    benchmark_result_t res = { .runtime = { avg, med, min, max },
							   .error = error };
	this->results = res; // not using temporary variable res gives NVCC compiler segfault ...
	return this->results;
}

template<typename value_t>
bool Benchmark::verify(Grid<value_t, coord3> *reference, Grid<value_t, coord3> *other, double tol) {
    if(other->dimensions != reference->dimensions) {
        return false;
    }
    for(int x=0; x<other->dimensions.x; x++) {
        for(int y=0; y<other->dimensions.y; y++) {
            for(int z=0; z<other->dimensions.z; z++) {
                /* The reason the benchmark times slow down this much if we use --no-verify flag
                is because comparing to the reference throws the values that the kernel needs out
                of the cache. The following proves that; i.e. not accessing the reference benchmarks
                keeps the other benchmark in cache and as such the next kernel run will be faster.
                */if(abs((*other)[coord3(x, y, z)]) > 1) {
                    return true;
                    //this->error = true;
                    //continue;
                }
                /*
                if(abs((*other)[coord3(x, y, z)] - (*reference)[coord3(x, y, z)]) > tol) {
                    return false;
                }*/
            }
        }
    }
    return true;
}

void Benchmark::parse_args() {
}

#endif