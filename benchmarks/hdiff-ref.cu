#include "hip/hip_runtime.h"
#ifndef HDIFF_REF_H
#define HDIFF_REF_H
#include "benchmarks/benchmark.cu"
#include "coord3.cu"
#include "grids/grid.cu"
#include "grids/regular.cu"
#include "grids/unstructured.cu"

/** This is the reference implementation for the horizontal diffusion kernel, 
 * which is executed on the CPU and used to verify other implementations. */
template<typename value_t>
class HdiffReferenceBenchmark : public Benchmark {

    public:

    // The padding option currently only applies to regular grids
    HdiffReferenceBenchmark(coord3 size);
 
    RegularGrid3D<value_t> *input = NULL;
    RegularGrid3D<value_t> *output = NULL;
    RegularGrid3D<value_t> *coeff = NULL;
    RegularGrid3D<value_t> *lap = NULL;
    RegularGrid3D<value_t> *flx = NULL;
    RegularGrid3D<value_t> *fly = NULL;

    // Setup Input values
    // As in hdiff_stencil_variant.h
    virtual void setup();
    virtual void populate_grids();
    virtual void teardown();
    virtual bool setup_from_archive(Benchmark::cache_iarchive &ar);
    virtual void store_to_archive(Benchmark::cache_oarchive &ar);

    // CPU implementation
    // As in hdiff_stencil_variant.h
    void run();

    // halo around the input data, padding that is not touched
    size_t padding;
    coord3 halo;
    coord3 inner_size; // size w.o. 2* halo

    // Print (1, 1, 1) for numblocks/numthreads as this is on CPU
    dim3 numblocks(coord3 domain);
    dim3 numthreads(coord3 domain);

};

// IMPLEMENTATIONS

template<typename value_t>
HdiffReferenceBenchmark<value_t>::HdiffReferenceBenchmark(coord3 size) :
Benchmark(size),
padding(padding),
halo(coord3(2,2,0)){
    this->name = "hdiff-ref";
}

template<typename value_t>
void HdiffReferenceBenchmark<value_t>::setup(){
    // Algorithm requires a halo: padding that is not touched
    this->inner_size = this->size - 2*this->halo;

    if(!this->setup_from_cache()) {
        // Set up grids
        this->input = RegularGrid3D<value_t>::create(this->inner_size, this->halo);
        this->output = RegularGrid3D<value_t>::create(this->inner_size, this->halo);
        this->coeff = RegularGrid3D<value_t>::create(this->inner_size, this->halo);
        this->lap = RegularGrid3D<value_t>::create(this->inner_size, this->halo);
        this->flx = RegularGrid3D<value_t>::create(this->inner_size, this->halo);
        this->fly = RegularGrid3D<value_t>::create(this->inner_size, this->halo);

        // Populate with data
        this->populate_grids();
        // do NOT call base setup here, this would lead to infinite recursion as
        // in base setup we create a reference benchmark such as this one
        this->store_to_cache();
    }
}

template<typename value_t>
bool HdiffReferenceBenchmark<value_t>::setup_from_archive(Benchmark::cache_iarchive &ar) {
    auto input = new RegularGrid3D<value_t>();
    auto output = new RegularGrid3D<value_t>();
    auto coeff = new RegularGrid3D<value_t>();
    auto lap = new RegularGrid3D<value_t>();
    auto flx = new RegularGrid3D<value_t>();
    auto fly = new RegularGrid3D<value_t>();
    ar >> *input;
    ar >> *output;
    ar >> *coeff;
    ar >> *lap;
    ar >> *flx;
    ar >> *fly;
    this->input = input;
    this->output = output;
    this->coeff = coeff;
    this->lap = lap;
    this->flx = flx;
    this->fly = fly;
    return true;
}

template<typename value_t>
void HdiffReferenceBenchmark<value_t>::store_to_archive(Benchmark::cache_oarchive &ar) {
    auto input = dynamic_cast<RegularGrid3D<value_t> *>(this->input);
    auto output = dynamic_cast<RegularGrid3D<value_t> *>(this->output);
    auto coeff = dynamic_cast<RegularGrid3D<value_t> *>(this->coeff);
    auto lap = dynamic_cast<RegularGrid3D<value_t> *>(this->lap);
    auto flx = dynamic_cast<RegularGrid3D<value_t> *>(this->flx);
    auto fly = dynamic_cast<RegularGrid3D<value_t> *>(this->fly);
    ar << *input;
    ar << *output;
    ar << *coeff;
    ar << *lap;
    ar << *flx;
    ar << *fly;
}

template<typename value_t>
void HdiffReferenceBenchmark<value_t>::populate_grids() {
    // Populate memory with values as in reference implementation (copied 1:1)
    value_t *m_in = this->input->data;
    value_t *m_out = this->output->data;
    value_t *m_coeff = this->coeff->data;
    value_t *m_lap = this->lap->data;
    value_t *m_flx = this->flx->data;
    value_t *m_fly = this->fly->data;
    const int isize = this->inner_size.x;
    const int jsize = this->inner_size.y;
    const int ksize = this->inner_size.z;
    // original code starts here
    value_t dx = 1. / (value_t)(isize);
    value_t dy = 1. / (value_t)(jsize);
    value_t dz = 1. / (value_t)(ksize);
    for (int j = 0; j < isize; j++) {
        for (int i = 0; i < jsize; i++) {
            value_t x = dx * (value_t)(i);
            value_t y = dy * (value_t)(j);
            for (int k = 0; k < ksize; k++) {
                int cnt = this->input->index(coord3(j, i, k)); // MODIFIED
                value_t z = dz * (value_t)(k);
                // u values between 5 and 9
                m_in[cnt] = 3.0 +
                            1.25 * (2.5 + cos(M_PI * (18.4 * x + 20.3 * y)) +
                                        0.78 * sin(2 * M_PI * (18.4 * x + 20.3 * y) * z)) /
                                4.;
                m_coeff[cnt] = 1.4 +
                                0.87 * (0.3 + cos(M_PI * (1.4 * x + 2.3 * y)) +
                                            1.11 * sin(2 * M_PI * (1.4 * x + 2.3 * y) * z)) /
                                    4.;
                m_out[cnt] = 5.4;
                m_flx[cnt] = 0.0;
                m_fly[cnt] = 0.0;
                m_lap[cnt] = 0.0;
            }
        }
    }
}

template<typename value_t>
void HdiffReferenceBenchmark<value_t>::run() {
    // Grids
    value_t *in = this->input->data;
    value_t *coeff = this->coeff->data;
    value_t *out_ref = this->output->data;
    value_t *lap_ref = this->lap->data;
    value_t *flx_ref = this->flx->data;
    value_t *fly_ref = this->fly->data;
    // convenience variables
    const int isize = this->inner_size.x;
    const int jsize = this->inner_size.y;
    const int ksize = this->inner_size.z;
    auto index = [this](int x, int y, int z) { return this->input->index(coord3(x, y, z)); };
    // begin copied code
    for (int k = 0; k < ksize; ++k) {
        for (int j = -1; j < jsize + 1; ++j) {
            for (int i = -1; i < isize + 1; ++i) {
                lap_ref[index(i, j, k)] =
                    4 * in[index(i, j, k)] - (in[index(i - 1, j, k)] + in[index(i + 1, j, k)] +
                                                   in[index(i, j - 1, k)] + in[index(i, j + 1, k)]);
            }
        }
        for (int j = 0; j < jsize; ++j) {
            for (int i = -1; i < isize; ++i) {
                flx_ref[index(i, j, k)] = lap_ref[index(i + 1, j, k)] - lap_ref[index(i, j, k)];
                if (flx_ref[index(i, j, k)] * (in[index(i + 1, j, k)] - in[index(i, j, k)]) > 0)
                    flx_ref[index(i, j, k)] = 0.;
            }
        }
        for (int j = -1; j < jsize; ++j) {
            for (int i = 0; i < isize; ++i) {
                fly_ref[index(i, j, k)] = lap_ref[index(i, j + 1, k)] - lap_ref[index(i, j, k)];
                if (fly_ref[index(i, j, k)] * (in[index(i, j + 1, k)] - in[index(i, j, k)]) > 0)
                    fly_ref[index(i, j, k)] = 0.;
            }
        }
        for (int i = 0; i < isize; ++i) {
            for (int j = 0; j < jsize; ++j) {
                out_ref[index(i, j, k)] =
                    in[index(i, j, k)] -
                    coeff[index(i, j, k)] * (flx_ref[index(i, j, k)] - flx_ref[index(i - 1, j, k)] +
                                                  fly_ref[index(i, j, k)] - fly_ref[index(i, j - 1, k)]);
            }
        }
    }
}

template<typename value_t>
void HdiffReferenceBenchmark<value_t>::teardown() {
    delete this->input;
    delete this->output;
    delete this->coeff;
    delete this->lap;
    delete this->flx;
    delete this->fly;
}

template<typename value_t>
dim3 HdiffReferenceBenchmark<value_t>::numblocks(coord3 domain) {
    return dim3(1, 1, 1);
}

template<typename value_t>
dim3 HdiffReferenceBenchmark<value_t>::numthreads(coord3 domain) {
    return dim3(1, 1, 1);
}

#endif