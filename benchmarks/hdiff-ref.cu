#include "hip/hip_runtime.h"
#ifndef HDIFF_REF_H
#define HDIFF_REF_H
#include "benchmarks/benchmark.cu"
#include "coord3.cu"
#include "grids/grid.cu"
#include "grids/regular.cu"
#include "grids/unstructured.cu"

enum HdiffBenchmarkGrids {RegularGrid, UnstructuredGrid};

/** This is the reference implementation for the horizontal diffusion kernel, 
 * which is executed on the CPU and used to verify other implementations. */
class HdiffReferenceBenchmark :  public Benchmark<double> {

    public:

    enum HdiffBenchmarkGrids grid_type;

    // The padding option currently only applies to regular grids
    HdiffReferenceBenchmark(coord3 size, enum HdiffBenchmarkGrids grid_type=RegularGrid, size_t padding=0);

    // additional grids besides input and output (which are defined in benchmark.h)
    Grid<double, coord3> *coeff;
    Grid<double, coord3> *lap;
    Grid<double, coord3> *flx;
    Grid<double, coord3> *fly;
 
    // Setup Input values
    // As in hdiff_stencil_variant.h
    virtual void setup();
    virtual void populate_grids();
    virtual void teardown();
    void post();

    // CPU implementation
    // As in hdiff_stencil_variant.h
    virtual void run();
    void calc_ref();
    void calc_ref_neighbors_rel();

    // halo around the input data, padding that is not touched
    size_t padding;
    coord3 halo;
    coord3 inner_size; // size w.o. 2* halo
    coord3 inner_coord(coord3 inner_coord);

};

// IMPLEMENTATIONS

HdiffReferenceBenchmark::HdiffReferenceBenchmark(coord3 size, enum HdiffBenchmarkGrids type, size_t padding)
: Benchmark(size),
  grid_type(type),
  padding(padding),
  halo(coord3(2,2,0)) {
    if(this->grid_type == RegularGrid) {
        if(this->padding > 0) {
            this->name = "hdiff-ref-pad";
        } else {
            this->name = "hdiff-ref";
        }
    } else {
        this->name = "hdiff-ref-unstr";
    }
}

void HdiffReferenceBenchmark::setup(){
    // Set up grids
    if(this->grid_type == RegularGrid) {
        this->input = new RegularGrid3D<double>(this->size, this->padding);
        this->output = new RegularGrid3D<double>(this->size, this->padding);
        this->coeff = new RegularGrid3D<double>(this->size, this->padding);
        this->lap = new RegularGrid3D<double>(this->size, this->padding);
        this->flx = new RegularGrid3D<double>(this->size, this->padding);
        this->fly = new RegularGrid3D<double>(this->size, this->padding);
    } else {
        this->input = UnstructuredGrid3D<double>::create_regular(this->size);
        this->output = UnstructuredGrid3D<double>::create_regular(this->size);
        this->coeff = UnstructuredGrid3D<double>::create_regular(this->size);
        this->lap = UnstructuredGrid3D<double>::create_regular(this->size);
        this->flx = UnstructuredGrid3D<double>::create_regular(this->size);
        this->fly = UnstructuredGrid3D<double>::create_regular(this->size);
    }
    // Algorithm requires a halo: padding that is not touched
    this->inner_size = this->size - 2*this->halo;
    // Populate with data
    this->populate_grids();
}

coord3 HdiffReferenceBenchmark::inner_coord(coord3 coord){
    return coord + this->halo;
}

void HdiffReferenceBenchmark::populate_grids() {
    // Populate memory with values as in reference implementation (copied 1:1)
    double *m_in = this->input->data;
    double *m_out = this->output->data;
    double *m_coeff = this->coeff->data;
    double *m_lap = this->lap->data;
    double *m_flx = this->flx->data;
    double *m_fly = this->fly->data;
    const int isize = this->inner_size.x;
    const int jsize = this->inner_size.y;
    const int ksize = this->inner_size.z;
    // original code starts here
    double dx = 1. / (double)(isize);
    double dy = 1. / (double)(jsize);
    double dz = 1. / (double)(ksize);
    for (int j = 0; j < isize; j++) {
        for (int i = 0; i < jsize; i++) {
            double x = dx * (double)(i);
            double y = dy * (double)(j);
            for (int k = 0; k < ksize; k++) {
                int cnt = this->input->index(this->inner_coord(coord3(j, i, k))); // MODIFIED
                double z = dz * (double)(k);
                // u values between 5 and 9
                m_in[cnt] = 3.0 +
                            1.25 * (2.5 + cos(M_PI * (18.4 * x + 20.3 * y)) +
                                        0.78 * sin(2 * M_PI * (18.4 * x + 20.3 * y) * z)) /
                                4.;
                m_coeff[cnt] = 1.4 +
                                0.87 * (0.3 + cos(M_PI * (1.4 * x + 2.3 * y)) +
                                            1.11 * sin(2 * M_PI * (1.4 * x + 2.3 * y) * z)) /
                                    4.;
                m_out[cnt] = 5.4;
                m_flx[cnt] = 0.0;
                m_fly[cnt] = 0.0;
                m_lap[cnt] = 0.0;
            }
        }
    }
}

void HdiffReferenceBenchmark::run() {
    if(this->grid_type == RegularGrid) {
        this->calc_ref();
    } else {
        this->calc_ref_neighbors_rel();
    }
    //this->output = this->flx; // DEBUG
}

void HdiffReferenceBenchmark::calc_ref() {
    const int isize = this->inner_size.x;
    const int jsize = this->inner_size.y;
    const int ksize = this->inner_size.z;
    double *in = this->input->data;
    double *out_ref = this->output->data;
    double *coeff = this->coeff->data;
    double *lap_ref = this->lap->data;
    double *flx_ref = this->flx->data;
    double *fly_ref = this->fly->data;
    auto index = [this](int x, int y, int z) { return this->input->index(this->inner_coord(coord3(x, y, z))); };
    // begin copied code
    for (int k = 0; k < ksize; ++k) {
        for (int j = -1; j < jsize + 1; ++j) {
            for (int i = -1; i < isize + 1; ++i) {
                lap_ref[index(i, j, k)] =
                    4 * in[index(i, j, k)] - (in[index(i - 1, j, k)] + in[index(i + 1, j, k)] +
                                                   in[index(i, j - 1, k)] + in[index(i, j + 1, k)]);
            }
        }
        for (int j = 0; j < jsize; ++j) {
            for (int i = -1; i < isize; ++i) {
                flx_ref[index(i, j, k)] = lap_ref[index(i + 1, j, k)] - lap_ref[index(i, j, k)];
                if (flx_ref[index(i, j, k)] * (in[index(i + 1, j, k)] - in[index(i, j, k)]) > 0)
                    flx_ref[index(i, j, k)] = 0.;
            }
        }
        for (int j = -1; j < jsize; ++j) {
            for (int i = 0; i < isize; ++i) {
                fly_ref[index(i, j, k)] = lap_ref[index(i, j + 1, k)] - lap_ref[index(i, j, k)];
                if (fly_ref[index(i, j, k)] * (in[index(i, j + 1, k)] - in[index(i, j, k)]) > 0)
                    fly_ref[index(i, j, k)] = 0.;
            }
        }
        for (int i = 0; i < isize; ++i) {
            for (int j = 0; j < jsize; ++j) {
                out_ref[index(i, j, k)] =
                    in[index(i, j, k)] -
                    coeff[index(i, j, k)] * (flx_ref[index(i, j, k)] - flx_ref[index(i - 1, j, k)] +
                                                  fly_ref[index(i, j, k)] - fly_ref[index(i, j - 1, k)]);
            }
        }
    }
}

// Same as calc ref, but uses neighbors relations instead of directly indexing
void HdiffReferenceBenchmark::calc_ref_neighbors_rel() {
    const int isize = this->inner_size.x;
    const int jsize = this->inner_size.y;
    const int ksize = this->inner_size.z;
    double *in = this->input->data;
    double *out_ref = this->output->data;
    double *coeff = this->coeff->data;
    double *lap_ref = this->lap->data;
    double *flx_ref = this->flx->data;
    double *fly_ref = this->fly->data;
    //auto index = [this](int x, int y, int z) { return this->input->index(this->inner_coord(coord3(x, y, z))); };
    // begin copied code
    for (int k = 0; k < ksize; ++k) {
        for (int j = -1; j < jsize + 1; ++j) {
            for (int i = -1; i < isize + 1; ++i) {
                coord3 cell = this->inner_coord(coord3(i, j, k));
                lap_ref[this->lap->index(cell)] =
                    4 * in[this->input->index(cell)] 
                    - (   in[this->input->neighbor(cell, coord3(-1, 0, 0))] 
                        + in[this->input->neighbor(cell, coord3(+1, 0, 0))]
                        + in[this->input->neighbor(cell, coord3(0, -1, 0))]
                        + in[this->input->neighbor(cell, coord3(0, +1, 0))]);
            }
        }
        for (int j = 0; j < jsize; ++j) {
            for (int i = -1; i < isize; ++i) {
                coord3 cell = this->inner_coord(coord3(i, j, k));
                flx_ref[this->flx->index(cell)] = 
                      lap_ref[this->lap->neighbor(cell, coord3(+1, 0, 0))]
                    - lap_ref[this->lap->index(cell)];
                if (flx_ref[this->lap->index(cell)]
                    * (  in[this->input->neighbor(cell, coord3(+1, 0, 0))] 
                       - in[this->input->index(cell)]) > 0)
                    flx_ref[this->flx->index(cell)] = 0.;
            }
        }
        for (int j = -1; j < jsize; ++j) {
            for (int i = 0; i < isize; ++i) {
                coord3 cell = this->inner_coord(coord3(i, j, k));
                fly_ref[this->fly->index(cell)] = 
                      lap_ref[this->lap->neighbor(cell, coord3(0, +1, 0))] 
                    - lap_ref[this->lap->index(cell)];
                if (fly_ref[this->fly->index(cell)] 
                    * (  in[this->input->neighbor(cell, coord3(0, +1, 0))] 
                       - in[this->input->index(cell)]) > 0)
                    fly_ref[this->fly->index(cell)] = 0.;
            }
        }
        for (int i = 0; i < isize; ++i) {
            for (int j = 0; j < jsize; ++j) {
                coord3 cell = this->inner_coord(coord3(i, j, k));
                out_ref[this->output->index(cell)] =
                      in[this->input->index(cell)]
                    - coeff[this->coeff->index(cell)]
                    * (  flx_ref[this->flx->index(cell)]
                       - flx_ref[this->flx->neighbor(cell, coord3(-1, 0, 0))]
                       + fly_ref[this->fly->index(cell)]
                       - fly_ref[this->fly->neighbor(cell, coord3(0, -1, 0))]);
            }
        }
    }
}

void HdiffReferenceBenchmark::post() {
    // in default implementation, we have cudaPeekAtError here, but that makes
    // no sense for this CPU implementation
    /*printf("\n==============\n%s\n", this->name.c_str());
    this->lap->print();
    printf("\n--- flx ---\n");
    this->flx->print();
    printf("\n--- fly ---\n");
    this->fly->print();
    printf("\n--- out --- \n");
    this->output->print();*/
}

void HdiffReferenceBenchmark::teardown() {

}

#endif