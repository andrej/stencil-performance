#include "hip/hip_runtime.h"
#ifndef HDIFF_REF_H
#define HDIFF_REF_H
#include "benchmarks/benchmark.cu"
#include "coord3.cu"
#include "grids/grid.cu"
#include "grids/regular.cu"
#include "grids/unstructured.cu"

/** This is the reference implementation for the horizontal diffusion kernel, 
 * which is executed on the CPU and used to verify other implementations. */
class HdiffReferenceBenchmark : public Benchmark<double> {

    public:

    // The padding option currently only applies to regular grids
    HdiffReferenceBenchmark(coord3 size, size_t padding=0);
 
    RegularGrid3D<double> *coeff = NULL;
    RegularGrid3D<double> *lap = NULL;
    RegularGrid3D<double> *flx = NULL;
    RegularGrid3D<double> *fly = NULL;

    // Setup Input values
    // As in hdiff_stencil_variant.h
    virtual void setup();
    virtual void populate_grids();
    virtual void teardown();

    // CPU implementation
    // As in hdiff_stencil_variant.h
    void run();

    // halo around the input data, padding that is not touched
    size_t padding;
    coord3 halo;
    coord3 inner_size; // size w.o. 2* halo
    coord3 inner_coord(coord3 inner_coord);

    // Print (1, 1, 1) for numblocks/numthreads as this is on CPU
    dim3 numblocks();
    dim3 numthreads();

};

// IMPLEMENTATIONS

HdiffReferenceBenchmark::HdiffReferenceBenchmark(coord3 size, size_t padding) :
Benchmark<double>(size),
padding(padding),
halo(coord3(2,2,0)){
    if(this->padding > 0) {
        this->name = "hdiff-ref-pad";
    } else {
        this->name = "hdiff-ref";
    }
}

void HdiffReferenceBenchmark::setup(){
    // Set up grids
    this->input = new RegularGrid3D<double>(this->size, this->padding);
    this->output = new RegularGrid3D<double>(this->size, this->padding);
    this->coeff = new RegularGrid3D<double>(this->size, this->padding);
    this->lap = new RegularGrid3D<double>(this->size, this->padding);
    this->flx = new RegularGrid3D<double>(this->size, this->padding);
    this->fly = new RegularGrid3D<double>(this->size, this->padding);
    // Algorithm requires a halo: padding that is not touched
    this->inner_size = this->size - 2*this->halo;
    // Populate with data
    this->populate_grids();
    // do NOT call base setup here, this would lead to infinite recursion as
    // in base setup we create a reference benchmark such as this one
}

void HdiffReferenceBenchmark::populate_grids() {
    // Populate memory with values as in reference implementation (copied 1:1)
    double *m_in = this->input->data;
    double *m_out = this->output->data;
    double *m_coeff = this->coeff->data;
    double *m_lap = this->lap->data;
    double *m_flx = this->flx->data;
    double *m_fly = this->fly->data;
    const int isize = this->inner_size.x;
    const int jsize = this->inner_size.y;
    const int ksize = this->inner_size.z;
    // original code starts here
    double dx = 1. / (double)(isize);
    double dy = 1. / (double)(jsize);
    double dz = 1. / (double)(ksize);
    for (int j = 0; j < isize; j++) {
        for (int i = 0; i < jsize; i++) {
            double x = dx * (double)(i);
            double y = dy * (double)(j);
            for (int k = 0; k < ksize; k++) {
                int cnt = this->input->index(this->inner_coord(coord3(j, i, k))); // MODIFIED
                double z = dz * (double)(k);
                // u values between 5 and 9
                m_in[cnt] = 3.0 +
                            1.25 * (2.5 + cos(M_PI * (18.4 * x + 20.3 * y)) +
                                        0.78 * sin(2 * M_PI * (18.4 * x + 20.3 * y) * z)) /
                                4.;
                m_coeff[cnt] = 1.4 +
                                0.87 * (0.3 + cos(M_PI * (1.4 * x + 2.3 * y)) +
                                            1.11 * sin(2 * M_PI * (1.4 * x + 2.3 * y) * z)) /
                                    4.;
                m_out[cnt] = 5.4;
                m_flx[cnt] = 0.0;
                m_fly[cnt] = 0.0;
                m_lap[cnt] = 0.0;
            }
        }
    }
}

void HdiffReferenceBenchmark::run() {
    // Grids
    double *in = this->input->data;
    double *coeff = this->coeff->data;
    double *out_ref = this->output->data;
    double *lap_ref = this->lap->data;
    double *flx_ref = this->flx->data;
    double *fly_ref = this->fly->data;
    // convenience variables
    const int isize = this->inner_size.x;
    const int jsize = this->inner_size.y;
    const int ksize = this->inner_size.z;
    auto index = [this](int x, int y, int z) { return this->input->index(this->inner_coord(coord3(x, y, z))); };
    // begin copied code
    for (int k = 0; k < ksize; ++k) {
        for (int j = -1; j < jsize + 1; ++j) {
            for (int i = -1; i < isize + 1; ++i) {
                lap_ref[index(i, j, k)] =
                    4 * in[index(i, j, k)] - (in[index(i - 1, j, k)] + in[index(i + 1, j, k)] +
                                                   in[index(i, j - 1, k)] + in[index(i, j + 1, k)]);
            }
        }
        for (int j = 0; j < jsize; ++j) {
            for (int i = -1; i < isize; ++i) {
                flx_ref[index(i, j, k)] = lap_ref[index(i + 1, j, k)] - lap_ref[index(i, j, k)];
                if (flx_ref[index(i, j, k)] * (in[index(i + 1, j, k)] - in[index(i, j, k)]) > 0)
                    flx_ref[index(i, j, k)] = 0.;
            }
        }
        for (int j = -1; j < jsize; ++j) {
            for (int i = 0; i < isize; ++i) {
                fly_ref[index(i, j, k)] = lap_ref[index(i, j + 1, k)] - lap_ref[index(i, j, k)];
                if (fly_ref[index(i, j, k)] * (in[index(i, j + 1, k)] - in[index(i, j, k)]) > 0)
                    fly_ref[index(i, j, k)] = 0.;
            }
        }
        for (int i = 0; i < isize; ++i) {
            for (int j = 0; j < jsize; ++j) {
                out_ref[index(i, j, k)] =
                    in[index(i, j, k)] -
                    coeff[index(i, j, k)] * (flx_ref[index(i, j, k)] - flx_ref[index(i - 1, j, k)] +
                                                  fly_ref[index(i, j, k)] - fly_ref[index(i, j - 1, k)]);
            }
        }
    }
}

void HdiffReferenceBenchmark::teardown() {
    // Don't free, because this reference benchmark will be reused.
    //this->input->deallocate();
    //this->output->deallocate();
    //this->coeff->deallocate();
    //this->lap->deallocate();
    //this->flx->deallocate();
    //this->fly->deallocate();
}

coord3 HdiffReferenceBenchmark::inner_coord(coord3 coord){
    return coord + this->halo;
}

dim3 HdiffReferenceBenchmark::numblocks() {
    return dim3(1, 1, 1);
}

dim3 HdiffReferenceBenchmark::numthreads() {
    return dim3(1, 1, 1);
}

#endif